#include "hip/hip_runtime.h"
// -*-c++-*-
#ifdef STORMM_USE_CUDA
#  include <hip/hip_runtime.h>
#endif
#include "copyright.h"
#include "Accelerator/ptx_macros.h"
#include "Numerics/split_fixed_precision.h"
#include "map_density.h"
#include "pmigrid.h"

namespace stormm {
namespace energy {

#include "Accelerator/syncwarp.cui"
#include "Math/bspline.cui"
#include "Numerics/accumulation.cui"
#include "cellgrid_imaging.cui"

// Compile the __shared__ memory density accumulation kernels.  The format of each name is
// "kSA" + {l,s} + {i, r} + {d,f} + [4, 6] + {d,s} + "MapDensity".  The {l,s} branch indicates
// whether the coordinates in the cell grid have a short (32-bit) or long (64-bit) representation.
// The {i,r} branch indicates whether the coordinate rpresentation is real [r] or fixed-precision
// integer [i].  The final {d,f} branch indicates whether the calculations are to be performed in
// double- or single-precision, and is followed by the interpolation order (each order must get its
// own kernel in the interest of register conservation).  The final {d,s} branch indicates whether
// to carry out accumulations in double (95-bit) or single (63-bit) accumulations, necessary to
// reduce register pressure hat would otherwise be incurred by combining branches of the innermost
// loop and to take all accumulation buffers into __shared__.  Begin with the double-precision
// kernels.
#define ACC_MODE_DOUBLE
#define CALC_MODE_DOUBLE
#define TCALC double
#define TCALC2 double2
#  define TMAT int
#  define T4 int4
#    define ORDER 4
#      define KERNEL_NAME kSAsid4dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAsid5dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAsid6dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT
#  define TMAT llint
#  define T4 llint4
#  define TMAT_IS_LONG 
#    define ORDER 4
#      define KERNEL_NAME kSAlid4dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAlid5dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAlid6dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef TMAT_IS_LONG 
#  undef T4
#  undef TMAT
#  define TMAT_IS_REAL
#  define TMAT float
#  define T4 float4
#    define ORDER 4
#      define KERNEL_NAME kSAsrd4dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAsrd5dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAsrd6dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT
#  define TMAT double
#  define T4 double4
#  define TMAT_IS_LONG 
#    define ORDER 4
#      define KERNEL_NAME kSAlrd4dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAlrd5dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAlrd6dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef TMAT_IS_LONG 
#  undef T4
#  undef TMAT
#  undef TMAT_IS_REAL
#undef TCALC2
#undef TCALC
#undef CALC_MODE_DOUBLE
#undef ACC_MODE_DOUBLE
  
#define CALC_MODE_DOUBLE
#define TCALC double
#define TCALC2 double2
#  define TMAT int
#  define T4 int4
#    define ORDER 4
#      define KERNEL_NAME kSAsid4sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAsid5sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAsid6sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT
#  define TMAT llint
#  define T4 llint4
#  define TMAT_IS_LONG 
#    define ORDER 4
#      define KERNEL_NAME kSAlid4sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAlid5sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAlid6sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef TMAT_IS_LONG 
#  undef T4
#  undef TMAT
#  define TMAT_IS_REAL
#  define TMAT float
#  define T4 float4
#    define ORDER 4
#      define KERNEL_NAME kSAsrd4sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAsrd5sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAsrd6sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT
#  define TMAT double
#  define T4 double4
#  define TMAT_IS_LONG 
#    define ORDER 4
#      define KERNEL_NAME kSAlrd4sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAlrd5sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAlrd6sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#      undef DENSITY_SPREADING_THREADS
#    undef ORDER
#  undef TMAT_IS_LONG 
#  undef T4
#  undef TMAT
#  undef TMAT_IS_REAL
#undef TCALC2
#undef TCALC
#undef CALC_MODE_DOUBLE
  
// Define the single-precision __shared__ memory density accumulation kernels.
#define ACC_MODE_DOUBLE
#define TCALC float
#define TCALC2 float2
#  define TMAT int
#  define T4 int4
#    define ORDER 4
#      define KERNEL_NAME kSAsif4dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAsif5dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAsif6dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT
#  define TMAT llint
#  define T4 llint4
#  define TMAT_IS_LONG
#    define ORDER 4
#      define KERNEL_NAME kSAlif4dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAlif5dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAlif6dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef TMAT_IS_LONG
#  undef T4
#  undef TMAT
#  define TMAT_IS_REAL
#  define TMAT float
#  define T4 float4
#    define ORDER 4
#      define KERNEL_NAME kSAsrf4dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAsrf5dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAsrf6dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT
#  define TMAT double
#  define T4 double4
#  define TMAT_IS_LONG
#    define ORDER 4
#      define KERNEL_NAME kSAlrf4dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAlrf5dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAlrf6dMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef TMAT_IS_LONG
#  undef T4
#  undef TMAT
#  undef TMAT_IS_REAL
#undef TCALC2
#undef TCALC
#undef ACC_MODE_DOUBLE
  
#define TCALC float
#define TCALC2 float2
#  define TMAT int
#  define T4 int4
#    define ORDER 4
#      define KERNEL_NAME kSAsif4sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAsif5sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAsif6sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT
#  define TMAT llint
#  define T4 llint4
#  define TMAT_IS_LONG
#    define ORDER 4
#      define KERNEL_NAME kSAlif4sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAlif5sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAlif6sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef TMAT_IS_LONG
#  undef T4
#  undef TMAT
#  define TMAT_IS_REAL
#  define TMAT float
#  define T4 float4
#    define ORDER 4
#      define KERNEL_NAME kSAsrf4sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAsrf5sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAsrf6sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT
#  define TMAT double
#  define T4 double4
#  define TMAT_IS_LONG
#    define ORDER 4
#      define KERNEL_NAME kSAlrf4sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAlrf5sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAlrf6sMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef TMAT_IS_LONG
#  undef T4
#  undef TMAT
#  undef TMAT_IS_REAL
#undef TCALC2
#undef TCALC

// Define additional __shared__ accumulation kernels for use when the accumulation can be
// expected to stay within one 32- or 64-bit accumulator, not requiring the overflow bits.
#define SHORT_FORMAT_ACCUMULATION
#define ACC_MODE_DOUBLE
#define CALC_MODE_DOUBLE
#define TCALC double
#define TCALC2 double2
#  define TMAT int
#  define T4 int4
#    define ORDER 4
#      define KERNEL_NAME kSAsid4dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAsid5dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAsid6dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT
#  define TMAT llint
#  define T4 llint4
#  define TMAT_IS_LONG 
#    define ORDER 4
#      define KERNEL_NAME kSAlid4dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAlid5dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAlid6dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef TMAT_IS_LONG 
#  undef T4
#  undef TMAT
#  define TMAT_IS_REAL
#  define TMAT float
#  define T4 float4
#    define ORDER 4
#      define KERNEL_NAME kSAsrd4dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAsrd5dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAsrd6dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT
#  define TMAT double
#  define T4 double4
#  define TMAT_IS_LONG 
#    define ORDER 4
#      define KERNEL_NAME kSAlrd4dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAlrd5dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAlrd6dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef TMAT_IS_LONG 
#  undef T4
#  undef TMAT
#  undef TMAT_IS_REAL
#undef TCALC2
#undef TCALC
#undef CALC_MODE_DOUBLE
#undef ACC_MODE_DOUBLE
  
#define CALC_MODE_DOUBLE
#define TCALC double
#define TCALC2 double2
#  define TMAT int
#  define T4 int4
#    define ORDER 4
#      define KERNEL_NAME kSAsid4ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAsid5ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAsid6ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT
#  define TMAT llint
#  define T4 llint4
#  define TMAT_IS_LONG 
#    define ORDER 4
#      define KERNEL_NAME kSAlid4ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAlid5ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAlid6ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef TMAT_IS_LONG 
#  undef T4
#  undef TMAT
#  define TMAT_IS_REAL
#  define TMAT float
#  define T4 float4
#    define ORDER 4
#      define KERNEL_NAME kSAsrd4ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAsrd5ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAsrd6ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT
#  define TMAT double
#  define T4 double4
#  define TMAT_IS_LONG 
#    define ORDER 4
#      define KERNEL_NAME kSAlrd4ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAlrd5ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAlrd6ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef TMAT_IS_LONG 
#  undef T4
#  undef TMAT
#  undef TMAT_IS_REAL
#undef TCALC2
#undef TCALC
#undef CALC_MODE_DOUBLE
  
// Define the single-precision __shared__ memory density accumulation kernels.
#define ACC_MODE_DOUBLE
#define TCALC float
#define TCALC2 float2
#  define TMAT int
#  define T4 int4
#    define ORDER 4
#      define KERNEL_NAME kSAsif4dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAsif5dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAsif6dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT
#  define TMAT llint
#  define T4 llint4
#  define TMAT_IS_LONG
#    define ORDER 4
#      define KERNEL_NAME kSAlif4dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAlif5dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAlif6dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef TMAT_IS_LONG
#  undef T4
#  undef TMAT
#  define TMAT_IS_REAL
#  define TMAT float
#  define T4 float4
#    define ORDER 4
#      define KERNEL_NAME kSAsrf4dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAsrf5dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAsrf6dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT
#  define TMAT double
#  define T4 double4
#  define TMAT_IS_LONG
#    define ORDER 4
#      define KERNEL_NAME kSAlrf4dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAlrf5dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAlrf6dsfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef TMAT_IS_LONG
#  undef T4
#  undef TMAT
#  undef TMAT_IS_REAL
#undef TCALC2
#undef TCALC
#undef ACC_MODE_DOUBLE
  
#define TCALC float
#define TCALC2 float2
#  define TMAT int
#  define T4 int4
#    define ORDER 4
#      define KERNEL_NAME kSAsif4ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAsif5ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAsif6ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT
#  define TMAT llint
#  define T4 llint4
#  define TMAT_IS_LONG
#    define ORDER 4
#      define KERNEL_NAME kSAlif4ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAlif5ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAlif6ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef TMAT_IS_LONG
#  undef T4
#  undef TMAT
#  define TMAT_IS_REAL
#  define TMAT float
#  define T4 float4
#    define ORDER 4
#      define KERNEL_NAME kSAsrf4ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAsrf5ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAsrf6ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT
#  define TMAT double
#  define T4 double4
#  define TMAT_IS_LONG
#    define ORDER 4
#      define KERNEL_NAME kSAlrf4ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME kSAlrf5ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME kSAlrf6ssfMapDensity
#        include "map_density_shracc.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef TMAT_IS_LONG
#  undef T4
#  undef TMAT
#  undef TMAT_IS_REAL
#undef TCALC2
#undef TCALC
#undef SHORT_FORMAT_ACCUMULATION

//-------------------------------------------------------------------------------------------------
extern hipFuncAttributes queryShrAccQMapKernelRequirements(const PrecisionModel calc_prec,
                                                            const PrecisionModel acc_prec,
                                                            const bool overflow_needed,
                                                            const size_t cg_tmat,
                                                            const int order) {
  hipFuncAttributes result;
  switch (calc_prec) {
  case PrecisionModel::DOUBLE:
    switch (acc_prec) {
    case PrecisionModel::DOUBLE:
      if (overflow_needed) {
        if (cg_tmat == int_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsid4dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsid4dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsid5dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsid5dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsid6dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsid6dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == llint_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlid4dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlid4dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlid5dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlid5dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlid6dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlid6dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == float_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrd4dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrd4dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrd5dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrd5dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrd6dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrd6dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == double_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrd4dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrd4dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrd5dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrd5dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrd6dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrd6dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
      }
      else {
        if (cg_tmat == int_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsid4dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsid4dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsid5dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsid5dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsid6dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsid6dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == llint_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlid4dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlid4dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlid5dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlid5dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlid6dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlid6dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == float_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrd4dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrd4dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrd5dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrd5dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrd6dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrd6dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == double_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrd4dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrd4dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrd5dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrd5dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrd6dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrd6dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
      }
      break;
    case PrecisionModel::SINGLE:
      if (overflow_needed) {
        if (cg_tmat == int_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsid4sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsid4sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsid5sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsid5sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsid6sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsid6sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == llint_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlid4sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlid4sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlid5sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlid5sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlid6sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlid6sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == float_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrd4sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrd4sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrd5sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrd5sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrd6sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrd6sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == double_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrd4sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrd4sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrd5sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrd5sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrd6sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrd6sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
      }
      else {
        if (cg_tmat == int_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsid4ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsid4ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsid5ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsid5ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsid6ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsid6ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == llint_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlid4ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlid4ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlid5ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlid5ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlid6ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlid6ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == float_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrd4ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrd4ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrd5ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrd5ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrd6ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrd6ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == double_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrd4ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrd4ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrd5ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrd5ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrd6ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrd6ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
      }
      break;
    }
    break;
  case PrecisionModel::SINGLE:
    switch (acc_prec) {
    case PrecisionModel::DOUBLE:
      if (overflow_needed) {
        if (cg_tmat == int_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsif4dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsif4dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsif5dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsif5dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsif6dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsif6dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == llint_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlif4dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlif4dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlif5dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlif5dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlif6dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlif6dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == float_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrf4dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrf4dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrf5dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrf5dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrf6dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrf6dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == double_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrf4dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrf4dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrf5dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrf5dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrf6dMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrf6dMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
      }
      else {
        if (cg_tmat == int_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsif4dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsif4dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsif5dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsif5dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsif6dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsif6dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == llint_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlif4dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlif4dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlif5dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlif5dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlif6dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlif6dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == float_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrf4dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrf4dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrf5dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrf5dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrf6dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrf6dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == double_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrf4dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrf4dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrf5dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrf5dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrf6dsfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrf6dsfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
      }
      break;
    case PrecisionModel::SINGLE:
      if (overflow_needed) {
        if (cg_tmat == int_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsif4sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsif4sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsif5sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsif5sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsif6sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsif6sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == llint_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlif4sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlif4sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlif5sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlif5sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlif6sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlif6sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == float_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrf4sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrf4sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrf5sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrf5sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrf6sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrf6sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == double_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrf4sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrf4sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrf5sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrf5sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrf6sMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrf6sMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
      }
      else {
        if (cg_tmat == int_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsif4ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsif4ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsif5ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsif5ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsif6ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsif6ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == llint_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlif4ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlif4ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlif5ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlif5ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlif6ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlif6ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == float_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrf4ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrf4ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrf5ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrf5ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAsrf6ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAsrf6ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
        else if (cg_tmat == double_type_index) {
          if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrf4ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrf4ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 5 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrf5ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrf5ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
          else if (order == 6 &&
                   hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kSAlrf6ssfMapDensity)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kSAlrf6ssfMapDensity.",
                  "queryShrAccQMapKernelRequirements");
          }
        }
      }
      break;
    }
    break;
  }
  return result;
}

// Compile the double-precision naive density mapping kernels.  The format of each name is
// "k" + {l,s} + {i, r} + {d,f} + [4, 6] + "MapDensity".  The {l,s} branch indicates whether the
// coordinates in the cell grid have a short (32-bit) or long (64-bit) representation.  The {i,r}
// branch indicates whether the coordinate rpresentation is real [r] or fixed-precision integer
// [i].  The final {d,f} branch indicates whether the calculations are to be performed in single-
// or double-precision.  The letter codes are followed by the interpolation order (each order must
// get its own interpolation order in the interest of register conservation).
#define TCALC double
#define TCALC2 double2
#define TCALC_IS_DOUBLE
#  define TMAT int
#  define T4 int4
#    define ORDER 4
#      define KERNEL_NAME ksid4MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME ksid5MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME ksid6MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT
#  define TMAT llint
#  define TMAT_IS_LONG
#  define T4 llint4
#    define ORDER 4
#      define KERNEL_NAME klid4MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME klid5MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME klid6MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT_IS_LONG
#  undef TMAT
#  define TMAT_IS_REAL
#  define TMAT float
#  define T4 float4
#    define ORDER 4
#      define KERNEL_NAME ksrd4MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME ksrd5MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME ksrd6MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT
#  define TMAT double
#  define TMAT_IS_LONG
#  define T4 double4
#    define ORDER 4
#      define KERNEL_NAME klrd4MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME klrd5MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME klrd6MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT_IS_LONG
#  undef TMAT
#  undef TMAT_IS_REAL
#undef TCALC_IS_DOUBLE
#undef TCALC2
#undef TCALC

// Compile the single-precision naive density mapping kernels
#define TCALC float
#define TCALC2 float2
#  define TMAT int
#  define T4 int4
#    define ORDER 4
#      define KERNEL_NAME ksif4MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME ksif5MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME ksif6MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT
#  define TMAT llint
#  define TMAT_IS_LONG
#  define T4 llint4
#    define ORDER 4
#      define KERNEL_NAME klif4MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME klif5MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME klif6MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT_IS_LONG
#  undef TMAT
#  define TMAT_IS_REAL
#  define TMAT float
#  define T4 float4
#    define ORDER 4
#      define KERNEL_NAME ksrf4MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME ksrf5MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME ksrf6MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT
#  define TMAT double
#  define TMAT_IS_LONG
#  define T4 double4
#    define ORDER 4
#      define KERNEL_NAME klrf4MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 5
#      define KERNEL_NAME klrf5MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#    define ORDER 6
#      define KERNEL_NAME klrf6MapDensity
#      include "map_density.cui"
#      undef KERNEL_NAME
#    undef ORDER
#  undef T4
#  undef TMAT_IS_LONG
#  undef TMAT
#  undef TMAT_IS_REAL
#undef TCALC2
#undef TCALC

//-------------------------------------------------------------------------------------------------
extern hipFuncAttributes queryGeneralQMapKernelRequirements(const PrecisionModel prec,
                                                             const size_t cg_tmat,
                                                             const int order) {
  hipFuncAttributes result;
  switch (prec) {
  case PrecisionModel::DOUBLE:
    if (cg_tmat == int_type_index) {
      if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(ksid4MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel ksid4MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
      else if (order == 5 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(ksid5MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel ksid5MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
      else if (order == 6 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(ksid6MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel ksid6MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
    }
    else if (cg_tmat == llint_type_index) {
      if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(klid4MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel klid4MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
      else if (order == 5 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(klid5MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel klid5MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
      else if (order == 6 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(klid6MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel klid6MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
    }
    else if (cg_tmat == float_type_index) {
      if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(ksrd4MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel ksrd4MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
      else if (order == 5 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(ksrd5MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel ksrd5MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
      else if (order == 6 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(ksrd6MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel ksrd6MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
    }
    else if (cg_tmat == double_type_index) {
      if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(klrd4MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel klrd4MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
      else if (order == 5 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(klrd5MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel klrd5MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
      else if (order == 6 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(klrd6MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel klrd6MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
    }
    break;
  case PrecisionModel::SINGLE:
    if (cg_tmat == int_type_index) {
      if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(ksif4MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel ksif4MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
      else if (order == 5 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(ksif5MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel ksif5MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
      else if (order == 6 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(ksif6MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel ksif6MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
    }
    else if (cg_tmat == llint_type_index) {
      if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(klif4MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel klif4MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
      else if (order == 5 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(klif5MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel klif5MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
      else if (order == 6 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(klif6MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel klif6MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
    }
    else if (cg_tmat == float_type_index) {
      if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(ksrf4MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel ksrf4MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
      else if (order == 5 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(ksrf5MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel ksrf5MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
      else if (order == 6 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(ksrf6MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel ksrf6MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
    }
    else if (cg_tmat == double_type_index) {
      if (order == 4 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(klrf4MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel klrf4MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
      else if (order == 5 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(klrf5MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel klrf5MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
      else if (order == 6 && hipFuncGetAttributes(&result, reinterpret_cast<const void*>(klrf6MapDensity)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel klrf6MapDensity.",
              "queryGeneralQMapKernelRequirements");
      }
    }
    break;
  }
  return result;
}  

//-------------------------------------------------------------------------------------------------
extern void launchShrAccDensityKernel(PMIGridWriter *pm_wrt, const bool overflow,
                                      MMControlKit<double> *ctrl,
                                      const CellGridReader<void, void, void, void> &v_cgr,
                                      const size_t cg_tmat,
                                      const SyNonbondedKit<double, double2> &synbk,
                                      const int2 lp) {
  matchThemes(pm_wrt->theme, v_cgr.theme);
  switch (pm_wrt->mode) {
  case PrecisionModel::DOUBLE:
    if (cg_tmat == int_type_index) {
      const CellGridReader<int, void, double, int4> cgr = restoreType<int, void,
                                                                      double, int4>(v_cgr);
      switch (pm_wrt->order) {
      case 4:
        if (overflow) kSAsid4dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsid4dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 5:
        if (overflow) kSAsid5dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsid5dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 6:
        if (overflow) kSAsid6dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsid6dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      default:
        break;
      }
    }
    else if (cg_tmat == llint_type_index) {
      const CellGridReader<llint, void, double, llint4> cgr = restoreType<llint, void,
                                                                          double, llint4>(v_cgr);
      switch (pm_wrt->order) {
      case 4:
        if (overflow) kSAlid4dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlid4dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 5:
        if (overflow) kSAlid5dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlid5dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 6:
        if (overflow) kSAlid6dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlid6dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      default:
        break;
      }
    }
    else if (cg_tmat == float_type_index) {
      const CellGridReader<float, void, double, float4> cgr = restoreType<float, void,
                                                                          double, float4>(v_cgr);
      switch (pm_wrt->order) {
      case 4:
        if (overflow) kSAsrd4dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsrd4dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 5:
        if (overflow) kSAsrd5dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsrd5dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 6:
        if (overflow) kSAsrd6dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsrd6dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      default:
        break;
      }
    }
    else if (cg_tmat == double_type_index) {
      const CellGridReader<double, void,
                           double, double4> cgr = restoreType<double, void,
                                                              double, double4>(v_cgr);
      switch (pm_wrt->order) {
      case 4:
        if (overflow) kSAlrd4dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlrd4dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 5:
        if (overflow) kSAlrd5dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlrd5dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 6:
        if (overflow) kSAlrd6dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlrd6dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      default:
        break;
      }
    }
    break;
  case PrecisionModel::SINGLE:
    if (cg_tmat == int_type_index) {
      const CellGridReader<int, void, double, int4> cgr = restoreType<int, void,
                                                                      double, int4>(v_cgr);
      switch (pm_wrt->order) {
      case 4:
        if (overflow) kSAsid4sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsid4ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 5:
        if (overflow) kSAsid5sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsid5ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 6:
        if (overflow) kSAsid6sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsid6ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      default:
        break;
      }
    }
    else if (cg_tmat == llint_type_index) {
      const CellGridReader<llint, void, double, llint4> cgr = restoreType<llint, void,
                                                                          double, llint4>(v_cgr);
      switch (pm_wrt->order) {
      case 4:
        if (overflow) kSAlid4sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlid4ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 5:
        if (overflow) kSAlid5sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlid5ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 6:
        if (overflow) kSAlid6sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlid6ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      default:
        break;
      }
    }
    else if (cg_tmat == float_type_index) {
      const CellGridReader<float, void, double, float4> cgr = restoreType<float, void,
                                                                          double, float4>(v_cgr);
      switch (pm_wrt->order) {
      case 4:
        if (overflow) kSAsrd4sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsrd4ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 5:
        if (overflow) kSAsrd5sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsrd5ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 6:
        if (overflow) kSAsrd6sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsrd6ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      default:
        break;
      }
    }
    else if (cg_tmat == double_type_index) {
      const CellGridReader<double, void,
                           double, double4> cgr = restoreType<double, void,
                                                              double, double4>(v_cgr);
      switch (pm_wrt->order) {
      case 4:
        if (overflow) kSAlrd4sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlrd4ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 5:
        if (overflow) kSAlrd5sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlrd5ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 6:
        if (overflow) kSAlrd6sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlrd6ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      default:
        break;
      }
    }
    break;
  }
}

//-------------------------------------------------------------------------------------------------
extern void launchShrAccDensityKernel(PMIGridWriter *pm_wrt, const bool overflow,
                                      MMControlKit<float> *ctrl,
                                      const CellGridReader<void, void, void, void> &v_cgr,
                                      const size_t cg_tmat,
                                      const SyNonbondedKit<float, float2> &synbk, const int2 lp) {
  matchThemes(pm_wrt->theme, v_cgr.theme);
  switch (pm_wrt->mode) {
  case PrecisionModel::DOUBLE:
    if (cg_tmat == int_type_index) {
      const CellGridReader<int, void, float, int4> cgr = restoreType<int, void,
                                                                     float, int4>(v_cgr);
      switch (pm_wrt->order) {
      case 4:
        if (overflow) kSAsif4dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsif4dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 5:
        if (overflow) kSAsif5dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsif5dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 6:
        if (overflow) kSAsif6dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsif6dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      default:
        break;
      }
    }
    else if (cg_tmat == llint_type_index) {
      const CellGridReader<llint, void, float, llint4> cgr = restoreType<llint, void,
                                                                         float, llint4>(v_cgr);
      switch (pm_wrt->order) {
      case 4:
        if (overflow) kSAlif4dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlif4dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 5:
        if (overflow) kSAlif5dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlif5dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 6:
        if (overflow) kSAlif6dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlif6dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      default:
        break;
      }
    }
    else if (cg_tmat == float_type_index) {
      const CellGridReader<float, void, float, float4> cgr = restoreType<float, void,
                                                                         float, float4>(v_cgr);
      switch (pm_wrt->order) {
      case 4:
        if (overflow) kSAsrf4dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsrf4dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 5:
        if (overflow) kSAsrf5dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsrf5dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 6:
        if (overflow) kSAsrf6dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsrf6dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      default:
        break;
      }
    }
    else if (cg_tmat == double_type_index) {
      const CellGridReader<double, void, float, double4> cgr = restoreType<double, void,
                                                                           float, double4>(v_cgr);
      switch (pm_wrt->order) {
      case 4:
        if (overflow) kSAlrf4dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlrf4dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 5:
        if (overflow) kSAlrf5dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlrf5dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 6:
        if (overflow) kSAlrf6dMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlrf6dsfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      default:
        break;
      }
    }
    break;
  case PrecisionModel::SINGLE:
    if (cg_tmat == int_type_index) {
      const CellGridReader<int, void, float, int4> cgr = restoreType<int, void,
                                                                     float, int4>(v_cgr);
      switch (pm_wrt->order) {
      case 4:
        if (overflow) kSAsif4sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsif4ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 5:
        if (overflow) kSAsif5sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsif5ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 6:
        if (overflow) kSAsif6sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsif6ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      default:
        break;
      }
    }
    else if (cg_tmat == llint_type_index) {
      const CellGridReader<llint, void, float, llint4> cgr = restoreType<llint, void,
                                                                         float, llint4>(v_cgr);
      switch (pm_wrt->order) {
      case 4:
        if (overflow) kSAlif4sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlif4ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 5:
        if (overflow) kSAlif5sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlif5ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 6:
        if (overflow) kSAlif6sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlif6ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      default:
        break;
      }
    }
    else if (cg_tmat == float_type_index) {
      const CellGridReader<float, void, float, float4> cgr = restoreType<float, void,
                                                                         float, float4>(v_cgr);
      switch (pm_wrt->order) {
      case 4:
        if (overflow) kSAsrf4sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsrf4ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 5:
        if (overflow) kSAsrf5sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsrf5ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 6:
        if (overflow) kSAsrf6sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAsrf6ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      default:
        break;
      }
    }
    else if (cg_tmat == double_type_index) {
      const CellGridReader<double, void, float, double4> cgr = restoreType<double, void,
                                                                           float, double4>(v_cgr);
      switch (pm_wrt->order) {
      case 4:
        if (overflow) kSAlrf4sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlrf4ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 5:
        if (overflow) kSAlrf5sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlrf5ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      case 6:
        if (overflow) kSAlrf6sMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        else kSAlrf6ssfMapDensity<<<lp.x, lp.y>>>(*pm_wrt, *ctrl, cgr, synbk);
        break;
      default:
        break;
      }
    }
    break;
  }
}

//-------------------------------------------------------------------------------------------------
extern void launchGenPrpDensityKernel(PMIGridAccumulator *pm_acc,
                                      const CellGridReader<void, void, void, void> &v_cgr,
                                      const size_t cg_tmat,
                                      const SyNonbondedKit<double, double2> &synbk,
                                      const int2 lp) {
  matchThemes(pm_acc->theme, v_cgr.theme);
  if (cg_tmat == int_type_index) {
    const CellGridReader<int, void, double, int4> cgr = restoreType<int, void,
                                                                    double, int4>(v_cgr);
    switch (pm_acc->order) {
    case 4:
      ksid4MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    case 5:
      ksid5MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    case 6:
      ksid6MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    default:
      break;
    }
  }
  else if (cg_tmat == llint_type_index) {
    const CellGridReader<llint, void, double, llint4> cgr = restoreType<llint, void,
                                                                        double, llint4>(v_cgr);
    switch (pm_acc->order) {
    case 4:
      klid4MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    case 5:
      klid5MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    case 6:
      klid6MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    default:
      break;
    }
  }
  else if (cg_tmat == float_type_index) {
    const CellGridReader<float, void, double, float4> cgr = restoreType<float, void,
                                                                        double, float4>(v_cgr);
    switch (pm_acc->order) {
    case 4:
      ksrd4MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    case 5:
      ksrd5MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    case 6:
      ksrd6MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    default:
      break;
    }
  }
  else if (cg_tmat == double_type_index) {
    const CellGridReader<double, void, double, double4> cgr = restoreType<double, void,
                                                                          double, double4>(v_cgr);
    switch (pm_acc->order) {
    case 4:
      klrd4MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    case 5:
      klrd5MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    case 6:
      klrd6MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    default:
      break;
    }
  }
}

//-------------------------------------------------------------------------------------------------
extern void launchGenPrpDensityKernel(PMIGridAccumulator *pm_acc,
                                      const CellGridReader<void, void, void, void> &v_cgr,
                                      const size_t cg_tmat,
                                      const SyNonbondedKit<float, float2> &synbk, const int2 lp) {
  matchThemes(pm_acc->theme, v_cgr.theme);
  if (cg_tmat == int_type_index) {
    const CellGridReader<int, void, float, int4> cgr = restoreType<int, void,
                                                                   float, int4>(v_cgr);
    switch (pm_acc->order) {
    case 4:
      ksif4MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    case 5:
      ksif5MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    case 6:
      ksif6MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    default:
      break;
    }
  }
  else if (cg_tmat == llint_type_index) {
    const CellGridReader<llint, void, float, llint4> cgr = restoreType<llint, void,
                                                                       float, llint4>(v_cgr);
    switch (pm_acc->order) {
    case 4:
      klif4MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    case 5:
      klif5MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    case 6:
      klif6MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    default:
      break;
    }
  }
  else if (cg_tmat == float_type_index) {
    const CellGridReader<float, void, float, float4> cgr = restoreType<float, void,
                                                                       float, float4>(v_cgr);
    switch (pm_acc->order) {
    case 4:
      ksrf4MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    case 5:
      ksrf5MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    case 6:
      ksrf6MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    default:
      break;
    }
  }
  else if (cg_tmat == double_type_index) {
    const CellGridReader<double, void, float, double4> cgr = restoreType<double, void,
                                                                         float, double4>(v_cgr);
    switch (pm_acc->order) {
    case 4:
      klrf4MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    case 5:
      klrf5MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    case 6:
      klrf6MapDensity<<<lp.x, lp.y>>>(*pm_acc, cgr, synbk);
      break;
    default:
      break;
    }
  }
}

} // namespace energy
} // namespace stormm
