#include "hip/hip_runtime.h"
// -*-c++-*-
#include "copyright.h"
#include "Accelerator/ptx_macros.h"
#include "Constants/hpc_bounds.h"
#include "Constants/scaling.h"
#include "DataTypes/common_types.h"
#include "DataTypes/stormm_vector_types.h"
#include "Numerics/split_fixed_precision.h"
#include "Potential/energy_abstracts.h"
#include "Random/random.h"
#include "Synthesis/implicit_solvent_workspace.h"
#include "Synthesis/nonbonded_workunit.h"
#include "Synthesis/synthesis_enumerators.h"
#include "Topology/atomgraph_enumerators.h"
#include "hpc_nonbonded_potential.h"

namespace stormm {
namespace energy {

using constants::PrecisionModel;
using constants::twice_warp_bits_mask_int;
using constants::twice_warp_size_int;
using constants::warp_size_int;
using constants::warp_bits;
using constants::warp_bits_mask_int;
using numerics::chooseAccumulationMethod;
using synthesis::NbwuKind;
using synthesis::small_block_max_imports;
using synthesis::small_block_max_atoms;
using synthesis::tile_groups_wu_abstract_length;
using topology::ImplicitSolventModel;
using namespace random;
  
//-------------------------------------------------------------------------------------------------
// Get the number atoms in a particular tile stretch.
//
// Arguments:
//   nbwu_map:  Details of the non-bonded work unit, condensed into a simple array of integers
//   pos:       Thread position in the list of atoms
//-------------------------------------------------------------------------------------------------
__device__ __forceinline__ int getTileSideAtomCount(const int* nbwu_map, const int pos) {
  const int key_idx  = pos / 4;
  const int key_slot = pos - (key_idx * 4);
  return ((nbwu_map[small_block_max_imports + 1 + key_idx] >> (8 * key_slot)) & 0xff);
}

#include "Math/rounding.cui"
#include "Numerics/accumulation.cui"
#include "Random/xor_shift_rng.cui"

//-------------------------------------------------------------------------------------------------
// Load coordinates relating to atoms in a non-bonded tile suitable for isolated boundary
// conditions.
//
// Overloaded:
//   - Work with an array of long long integers appropriate for single-precision arithmetic
//   - Work with dual arrays of long long int and int types, appropriate for double-precision
//     arithmetic
//
// Arguments:
//   pos:              Position in the tile list (not the atom list)
//   import_count:     Number of groups of atoms imported to populate one side of one or more tiles
//   iter:             Number of passes made by this or related routines (incrementation of
//                     iter is essential to maintain the correct procession through all loads)
//   nbwu_map:         Non-bonded work unit details            
//   read_crd:         Array of coordinates to read from
//   write_crd:        Array of coordinates to write into
//   read_crd_ovrf:    Overflow buffers for coordinates to be read
//   write_crd_ovrf:   Overflow buffers for local copies of coordinates
//   sh_tile_cog:      Array holding mean values of the positions of each imported atom group (this
//                     later expedites computing the center of geometry for each complete tile)
//   gpos_scale:       Scaling factor for coordinates in the fixed-precision representation (this
//                     is needed only to place dummy atom coordinates for blank slots of a tile)
//-------------------------------------------------------------------------------------------------
__device__ int loadTileCoordinates(const int pos, const int iter, const int* nbwu_map,
                                   const llint* read_crd, llint* write_crd, float* sh_tile_cog,
                                   const float gpos_scale) {
  const int tile_sides_per_warp = (warp_size_int / tile_length);
  const int warps_per_block = blockDim.x >> warp_bits;
  const int tile_lane_idx = (threadIdx.x & tile_length_bits_mask);
  const int import_count = nbwu_map[0];
  const int padded_import_count = devcRoundUp(import_count, tile_sides_per_warp);
  int rel_pos = pos - (iter * padded_import_count);
  while (rel_pos < padded_import_count) {
    float fval;
    if (rel_pos < import_count) {
      const size_t read_idx = nbwu_map[rel_pos + 1] + tile_lane_idx;
      const size_t write_idx = (rel_pos * tile_length) + tile_lane_idx;
      if (tile_lane_idx < getTileSideAtomCount(nbwu_map, rel_pos)) {
        const llint ival = __ldcs(&read_crd[read_idx]);
        fval = (float)(ival);
        write_crd[write_idx] = ival;
      }
      else {
        fval = (float)(0.0);
        write_crd[write_idx] = (128 * (rel_pos + 8) * tile_lane_idx) * gpos_scale;
      }
    }
    else {
      fval = (float)(0.0);
    }
    for (int i = half_tile_length; i > 0; i >>= 1) {
      fval += SHFL_DOWN(fval, i);
    }
    if (tile_lane_idx == 0 && rel_pos < import_count) {
      sh_tile_cog[rel_pos] = fval;
    }
    rel_pos += tile_sides_per_warp * warps_per_block;
  }
  return rel_pos + (iter * padded_import_count);
}

__device__ int loadTileCoordinates(const int pos, const int iter, const int* nbwu_map,
                                   const llint* read_crd, llint* write_crd,
                                   const int* read_crd_ovrf, int* write_crd_ovrf,
                                   double* sh_tile_cog, const double gpos_scale) {
  const int tile_sides_per_warp = (warp_size_int / tile_length);
  const int warps_per_block = blockDim.x >> warp_bits;
  const int tile_lane_idx = (threadIdx.x & tile_length_bits_mask);
  const int import_count = nbwu_map[0];
  const int padded_import_count = devcRoundUp(import_count, tile_sides_per_warp);
  int rel_pos = pos - (iter * padded_import_count);
  while (rel_pos < padded_import_count) {
    double fval;
    if (rel_pos < import_count) {
      const size_t read_idx = nbwu_map[rel_pos + 1] + tile_lane_idx;
      const size_t write_idx = (rel_pos * tile_length) + tile_lane_idx;
      if (tile_lane_idx < getTileSideAtomCount(nbwu_map, rel_pos)) {
        const llint ival = __ldcs(&read_crd[read_idx]);
        fval = (double)(ival);
        write_crd[write_idx] = ival;
        const int ival_ovrf = __ldcs(&read_crd_ovrf[read_idx]);
        fval += (double)(ival_ovrf) * max_llint_accumulation;
        write_crd_ovrf[write_idx] = ival_ovrf;
      }
      else {
        fval = 0.0;
        const int95_t fake_val = doubleToInt95((128 * (rel_pos + 8) * tile_lane_idx) * gpos_scale);
        write_crd[write_idx] = fake_val.x;
        write_crd_ovrf[write_idx] = fake_val.y;
      }
    }
    else {
      fval = 0.0;
    }
    for (int i = half_tile_length; i > 0; i >>= 1) {
      fval += SHFL_DOWN(fval, i);
    }
    if (tile_lane_idx == 0 && rel_pos < import_count) {
      sh_tile_cog[rel_pos] = fval;
    }
    rel_pos += tile_sides_per_warp * warps_per_block;
  }
  return rel_pos + (iter * padded_import_count);
}

//-------------------------------------------------------------------------------------------------
// Load scalar values (integral or real) from global memory into local arrays, on a tile-by-tile
// basis for non-bonded work units involving isolated systems with all-to-all interaction matrices.
//
// Overloaded:
//   - Copy the values directly
//   - Fold in a scalar multiple
//   - Fold in a scalar addition
//
// Parameter descriptors follow from loadTileCoordinates() above, with alterations:
//   read_array:   Generic array of (global) information to read from
//   write_array:  Generic (local) array of information to write 
//-------------------------------------------------------------------------------------------------
template <typename T> __device__
int loadTileProperty(const int pos, const int iter, const int* nbwu_map, const T* read_array,
                     T* write_array) {
  const int tile_sides_per_warp = (warp_size_int / tile_length);
  const int warps_per_block = blockDim.x >> warp_bits;
  const int tile_lane_idx = (threadIdx.x & tile_length_bits_mask);
  const int import_count = nbwu_map[0];
  const int padded_import_count = devcRoundUp(import_count, tile_sides_per_warp);  
  int rel_pos = pos - (iter * padded_import_count);
  while (rel_pos < padded_import_count) {
    if (rel_pos < import_count) {
      const size_t read_idx = nbwu_map[rel_pos + 1] + tile_lane_idx;
      const size_t write_idx = (rel_pos * tile_length) + tile_lane_idx;
      if (tile_lane_idx < getTileSideAtomCount(nbwu_map, rel_pos)) {
        write_array[write_idx] = __ldcs(&read_array[read_idx]);
      }
      else {
        write_array[write_idx] = (T)(0);
      }
    }
    rel_pos += tile_sides_per_warp * warps_per_block;
  }
  return rel_pos + (iter * padded_import_count);
}

template <typename T> __device__
int loadTileProperty(const int pos, const int iter, const int* nbwu_map, const T* read_array,
                     T* write_array, T multiplier) {
  const int tile_sides_per_warp = (warp_size_int / tile_length);
  const int warps_per_block = blockDim.x >> warp_bits;
  const int tile_lane_idx = (threadIdx.x & tile_length_bits_mask);
  const int import_count = nbwu_map[0];
  const int padded_import_count = devcRoundUp(import_count, tile_sides_per_warp);  
  int rel_pos = pos - (iter * padded_import_count);
  while (rel_pos < padded_import_count) {
    if (rel_pos < import_count) {
      const size_t read_idx = nbwu_map[rel_pos + 1] + tile_lane_idx;
      const size_t write_idx = (rel_pos * tile_length) + tile_lane_idx;
      if (tile_lane_idx < getTileSideAtomCount(nbwu_map, rel_pos)) {
        write_array[write_idx] = __ldcs(&read_array[read_idx]) * multiplier;
      }
      else {
        write_array[write_idx] = (T)(0);
      }
    }
    rel_pos += tile_sides_per_warp * warps_per_block;
  }
  return rel_pos + (iter * padded_import_count);
}

template <typename T> __device__
int loadTileProperty(const int pos, const int iter, const int* nbwu_map, const T* read_array,
                     T increment, T* write_array) {
  const int tile_sides_per_warp = (warp_size_int / tile_length);
  const int warps_per_block = blockDim.x >> warp_bits;
  const int tile_lane_idx = (threadIdx.x & tile_length_bits_mask);
  const int import_count = nbwu_map[0];
  const int padded_import_count = devcRoundUp(import_count, tile_sides_per_warp);  
  int rel_pos = pos - (iter * padded_import_count);
  while (rel_pos < padded_import_count) {
    if (rel_pos < import_count) {
      const size_t read_idx = nbwu_map[rel_pos + 1] + tile_lane_idx;
      const size_t write_idx = (rel_pos * tile_length) + tile_lane_idx;
      if (tile_lane_idx < getTileSideAtomCount(nbwu_map, rel_pos)) {
        write_array[write_idx] = __ldcs(&read_array[read_idx]) + increment;
      }
      else {
        write_array[write_idx] = (T)(0);
      }
    }
    rel_pos += tile_sides_per_warp * warps_per_block;
  }
  return rel_pos + (iter * padded_import_count);
}

//-------------------------------------------------------------------------------------------------
// Write information about the atoms in tile groups back to global accumulators.  Relevant for
// systems with all-to-all interactions in isolated boundary conditions.
//
// Overloaded:
//   - Accept various combinations of single- or double-integer local accumulators to contribute
//     to the implied single- or double-integer global accumulators
//
// Arguments:
//   pos:                   Position in the tile list (not the atom list)
//   iter:                  Number of passes made by this or related routines (incrementation of
//                          iter is essential to maintain the correct procession through all loads)
//   nbwu_map:              Non-bonded work unit details
//   tile_prop:             Primary local accumulator for the tile-based computed property
//   tile_prop_ovrf:        Local overflow accumulator for the tile-based computed property
//   gbl_accumulator:       Primary (or, perhaps lone) global accumulator for the computed property
//   gbl_accumulator_ovrf:  Overflow global accumulator for the computed property
//-------------------------------------------------------------------------------------------------
__device__ int accumulateTileProperty(const int pos, const int iter, const int* nbwu_map,
                                      const int* tile_prop, const int* tile_prop_ovrf,
                                      llint* gbl_accumulator) {
  const int tile_sides_per_warp = (warp_size_int / tile_length);
  const int warps_per_block = blockDim.x >> warp_bits;
  const int tile_lane_idx = (threadIdx.x & tile_length_bits_mask);
  const int import_count = nbwu_map[0];
  const int padded_import_count = devcRoundUp(import_count, tile_sides_per_warp);
  int rel_pos = pos - (iter * padded_import_count);
  while (rel_pos < padded_import_count) {
    if (rel_pos < import_count) {
      const size_t write_idx = nbwu_map[rel_pos + 1] + tile_lane_idx;
      const size_t read_idx = (rel_pos * tile_length) + tile_lane_idx;
      if (tile_lane_idx < getTileSideAtomCount(nbwu_map, rel_pos)) {
        llint itp = tile_prop_ovrf[read_idx];
        itp *= max_int_accumulation_ll;
        itp += tile_prop[read_idx];
        atomicAdd((ullint*)&gbl_accumulator[write_idx], (ullint)(itp));
      }
    }
    rel_pos += tile_sides_per_warp * warps_per_block;
  }
  return rel_pos + (iter * padded_import_count);
}

__device__ int accumulateTileProperty(const int pos, const int iter, const int* nbwu_map,
                                      const llint* tile_prop, llint* gbl_accumulator) {
  const int tile_sides_per_warp = (warp_size_int / tile_length);
  const int warps_per_block = blockDim.x >> warp_bits;
  const int tile_lane_idx = (threadIdx.x & tile_length_bits_mask);
  const int import_count = nbwu_map[0];
  const int padded_import_count = devcRoundUp(import_count, tile_sides_per_warp);
  int rel_pos = pos - (iter * padded_import_count);
  while (rel_pos < padded_import_count) {
    if (rel_pos < import_count) {
      const size_t write_idx = nbwu_map[rel_pos + 1] + tile_lane_idx;
      const size_t read_idx = (rel_pos * tile_length) + tile_lane_idx;
      if (tile_lane_idx < getTileSideAtomCount(nbwu_map, rel_pos)) {
        atomicAdd((ullint*)&gbl_accumulator[write_idx], (ullint)(tile_prop[read_idx]));
      }
    }
    rel_pos += tile_sides_per_warp * warps_per_block;
  }
  return rel_pos + (iter * padded_import_count);
}

__device__ int accumulateTileProperty(const int pos, const int iter, const int* nbwu_map,
                                      const llint* tile_prop, const int* tile_prop_ovrf,
                                      llint* gbl_accumulator, int* gbl_accumulator_ovrf) {
  const int tile_sides_per_warp = (warp_size_int / tile_length);
  const int warps_per_block = blockDim.x >> warp_bits;
  const int tile_lane_idx = (threadIdx.x & tile_length_bits_mask);
  const int import_count = nbwu_map[0];
  const int padded_import_count = devcRoundUp(import_count, tile_sides_per_warp);
  int rel_pos = pos - (iter * padded_import_count);
  while (rel_pos < padded_import_count) {
    if (rel_pos < import_count) {
      const size_t write_idx = nbwu_map[rel_pos + 1] + tile_lane_idx;
      const size_t read_idx = (rel_pos * tile_length) + tile_lane_idx;
      if (tile_lane_idx < getTileSideAtomCount(nbwu_map, rel_pos)) {
        atomicSplit(tile_prop[read_idx], tile_prop_ovrf[read_idx], write_idx, gbl_accumulator,
                    gbl_accumulator_ovrf);
      }
    }
    rel_pos += tile_sides_per_warp * warps_per_block;
  }
  return rel_pos + (iter * padded_import_count);
}

// Single-precision non-bonded kernel floating point definitions
#define TCALC float
#  define TCALC2 float2
#  define TCALC_IS_SINGLE
#  ifdef STORMM_USE_CUDA
#    if (__CUDA_ARCH__ >= 750) && (__CUDA_ARCH__ < 800)
#      define NONBOND_KERNEL_BLOCKS_MULTIPLIER 4
#      define GBRADII_KERNEL_BLOCKS_MULTIPLIER 4
#      define GBDERIV_KERNEL_BLOCKS_MULTIPLIER 4
#    else
#      define NONBOND_KERNEL_BLOCKS_MULTIPLIER 5
#      define GBRADII_KERNEL_BLOCKS_MULTIPLIER 5
#      define GBDERIV_KERNEL_BLOCKS_MULTIPLIER 5
#    endif
#  else
#    define NONBOND_KERNEL_BLOCKS_MULTIPLIER 5
#    define GBRADII_KERNEL_BLOCKS_MULTIPLIER 5
#    define GBDERIV_KERNEL_BLOCKS_MULTIPLIER 5
#  endif
#  define LLCONV_FUNC __float2ll_rn
#  define SQRT_FUNC sqrtf
#  define CBRT_FUNC cbrtf
#  define LOG_FUNC  logf
#  define EXP_FUNC  expf
#  define TANH_FUNC tanhf
#  define FABS_FUNC fabsf
#  define SPLIT_FORCE_ACCUMULATION
#    define KERNEL_NAME ktgfsCalculateGBRadii
#      include "gbradii_tilegroups.cui"
#    undef KERNEL_NAME
#    define DO_NECK_CORRECTION
#      define KERNEL_NAME ktgfsCalculateGBNeckRadii
#        include "gbradii_tilegroups.cui"
#      undef KERNEL_NAME
#    undef DO_NECK_CORRECTION
#  undef SPLIT_FORCE_ACCUMULATION
#  define KERNEL_NAME ktgfCalculateGBRadii
#    include "gbradii_tilegroups.cui"
#  undef KERNEL_NAME
#  define DO_NECK_CORRECTION
#    define KERNEL_NAME ktgfCalculateGBNeckRadii
#      include "gbradii_tilegroups.cui"
#    undef KERNEL_NAME
#  undef DO_NECK_CORRECTION
#  define COMPUTE_FORCE
#    define SPLIT_FORCE_ACCUMULATION
#      define COMPUTE_ENERGY
#        define KERNEL_NAME ktgfsVacuumForceEnergy
#          include "nonbonded_potential_tilegroups.cui"
#        undef KERNEL_NAME
#        define DO_GENERALIZED_BORN
#          define KERNEL_NAME ktgfsGBForceEnergy
#            include "nonbonded_potential_tilegroups.cui"
#          undef KERNEL_NAME
#          define DO_NECK_CORRECTION
#            define KERNEL_NAME ktgfsGBNeckForceEnergy
#              include "nonbonded_potential_tilegroups.cui"
#            undef KERNEL_NAME
#          undef DO_NECK_CORRECTION
#        undef DO_GENERALIZED_BORN
#      undef COMPUTE_ENERGY
#      define KERNEL_NAME ktgfsVacuumForce
#        include "nonbonded_potential_tilegroups.cui"
#      undef KERNEL_NAME
#      define DO_GENERALIZED_BORN
#        define KERNEL_NAME ktgfsGBForce
#          include "nonbonded_potential_tilegroups.cui"
#        undef KERNEL_NAME
#        define KERNEL_NAME ktgfsCalculateGBDerivatives
#          include "gbderivative_tilegroups.cui"
#        undef KERNEL_NAME
#        define DO_NECK_CORRECTION
#          define KERNEL_NAME ktgfsGBNeckForce
#            include "nonbonded_potential_tilegroups.cui"
#          undef KERNEL_NAME
#          define KERNEL_NAME ktgfsCalculateGBNeckDerivatives
#            include "gbderivative_tilegroups.cui"
#          undef KERNEL_NAME
#        undef DO_NECK_CORRECTION
#      undef DO_GENERALIZED_BORN
#    undef SPLIT_FORCE_ACCUMULATION
#    define COMPUTE_ENERGY
#      define KERNEL_NAME ktgfVacuumForceEnergy
#        include "nonbonded_potential_tilegroups.cui"
#      undef KERNEL_NAME
#      define DO_GENERALIZED_BORN
#        define KERNEL_NAME ktgfGBForceEnergy
#          include "nonbonded_potential_tilegroups.cui"
#        undef KERNEL_NAME
#        define DO_NECK_CORRECTION
#          define KERNEL_NAME ktgfGBNeckForceEnergy
#            include "nonbonded_potential_tilegroups.cui"
#          undef KERNEL_NAME
#        undef DO_NECK_CORRECTION
#      undef DO_GENERALIZED_BORN
#    undef COMPUTE_ENERGY
#    define KERNEL_NAME ktgfVacuumForce
#      include "nonbonded_potential_tilegroups.cui"
#    undef KERNEL_NAME
#    define DO_GENERALIZED_BORN
#      define KERNEL_NAME ktgfGBForce
#        include "nonbonded_potential_tilegroups.cui"
#      undef KERNEL_NAME
#      define KERNEL_NAME ktgfCalculateGBDerivatives
#        include "gbderivative_tilegroups.cui"
#      undef KERNEL_NAME
#      define DO_NECK_CORRECTION
#        define KERNEL_NAME ktgfGBNeckForce
#          include "nonbonded_potential_tilegroups.cui"
#        undef KERNEL_NAME
#        define KERNEL_NAME ktgfCalculateGBNeckDerivatives
#          include "gbderivative_tilegroups.cui"
#        undef KERNEL_NAME
#      undef DO_NECK_CORRECTION
#    undef DO_GENERALIZED_BORN
#  undef COMPUTE_FORCE
#  define COMPUTE_ENERGY
#    define KERNEL_NAME ktgfVacuumEnergy
#      include "nonbonded_potential_tilegroups.cui"
#    undef KERNEL_NAME
#    define DO_GENERALIZED_BORN
#      define KERNEL_NAME ktgfGBEnergy
#        include "nonbonded_potential_tilegroups.cui"
#      undef KERNEL_NAME
#      define DO_NECK_CORRECTION
#        define KERNEL_NAME ktgfGBNeckEnergy
#          include "nonbonded_potential_tilegroups.cui"
#        undef KERNEL_NAME
#      undef DO_NECK_CORRECTION
#    undef DO_GENERALIZED_BORN
#  undef COMPUTE_ENERGY
#  undef GBRADII_KERNEL_BLOCKS_MULTIPLIER
#  undef GBDERIV_KERNEL_BLOCKS_MULTIPLIER
#  undef NONBOND_KERNEL_BLOCKS_MULTIPLIER
#  define CLASH_FORGIVENESS
#    define COMPUTE_FORCE
#      define SPLIT_FORCE_ACCUMULATION
#        define COMPUTE_ENERGY
#          define NONBOND_KERNEL_BLOCKS_MULTIPLIER 4
#          define KERNEL_NAME ktgfsVacuumForceEnergyNonClash
#            include "nonbonded_potential_tilegroups.cui"
#          undef KERNEL_NAME
#          define DO_GENERALIZED_BORN
#            define KERNEL_NAME ktgfsGBForceEnergyNonClash
#              include "nonbonded_potential_tilegroups.cui"
#            undef KERNEL_NAME
#            define DO_NECK_CORRECTION
#              define KERNEL_NAME ktgfsGBNeckForceEnergyNonClash
#                include "nonbonded_potential_tilegroups.cui"
#              undef KERNEL_NAME
#            undef DO_NECK_CORRECTION
#          undef DO_GENERALIZED_BORN
#          undef NONBOND_KERNEL_BLOCKS_MULTIPLIER
#        undef COMPUTE_ENERGY
#        define NONBOND_KERNEL_BLOCKS_MULTIPLIER 5
#        define KERNEL_NAME ktgfsVacuumForceNonClash
#          include "nonbonded_potential_tilegroups.cui"
#        undef KERNEL_NAME
#        define DO_GENERALIZED_BORN
#          define KERNEL_NAME ktgfsGBForceNonClash
#            include "nonbonded_potential_tilegroups.cui"
#          undef KERNEL_NAME
#          define DO_NECK_CORRECTION
#            define KERNEL_NAME ktgfsGBNeckForceNonClash
#              include "nonbonded_potential_tilegroups.cui"
#            undef KERNEL_NAME
#          undef DO_NECK_CORRECTION
#        undef DO_GENERALIZED_BORN
#        undef NONBOND_KERNEL_BLOCKS_MULTIPLIER
#      undef SPLIT_FORCE_ACCUMULATION
#      define COMPUTE_ENERGY
#        define NONBOND_KERNEL_BLOCKS_MULTIPLIER 4
#        define KERNEL_NAME ktgfVacuumForceEnergyNonClash
#          include "nonbonded_potential_tilegroups.cui"
#        undef KERNEL_NAME
#        define DO_GENERALIZED_BORN
#          define KERNEL_NAME ktgfGBForceEnergyNonClash
#            include "nonbonded_potential_tilegroups.cui"
#          undef KERNEL_NAME
#          define DO_NECK_CORRECTION
#            define KERNEL_NAME ktgfGBNeckForceEnergyNonClash
#              include "nonbonded_potential_tilegroups.cui"
#            undef KERNEL_NAME
#          undef DO_NECK_CORRECTION
#        undef DO_GENERALIZED_BORN
#        undef NONBOND_KERNEL_BLOCKS_MULTIPLIER
#      undef COMPUTE_ENERGY
#      define NONBOND_KERNEL_BLOCKS_MULTIPLIER 5
#      define KERNEL_NAME ktgfVacuumForceNonClash
#        include "nonbonded_potential_tilegroups.cui"
#      undef KERNEL_NAME
#      define DO_GENERALIZED_BORN
#        define KERNEL_NAME ktgfGBForceNonClash
#          include "nonbonded_potential_tilegroups.cui"
#        undef KERNEL_NAME
#        define DO_NECK_CORRECTION
#          define KERNEL_NAME ktgfGBNeckForceNonClash
#            include "nonbonded_potential_tilegroups.cui"
#          undef KERNEL_NAME
#        undef DO_NECK_CORRECTION
#      undef DO_GENERALIZED_BORN
#      undef NONBOND_KERNEL_BLOCKS_MULTIPLIER
#    undef COMPUTE_FORCE
#    define COMPUTE_ENERGY
#      define NONBOND_KERNEL_BLOCKS_MULTIPLIER 5
#      define KERNEL_NAME ktgfVacuumEnergyNonClash
#        include "nonbonded_potential_tilegroups.cui"
#      undef KERNEL_NAME
#      define DO_GENERALIZED_BORN
#        define KERNEL_NAME ktgfGBEnergyNonClash
#          include "nonbonded_potential_tilegroups.cui"
#        undef KERNEL_NAME
#        define DO_NECK_CORRECTION
#          define KERNEL_NAME ktgfGBNeckEnergyNonClash
#            include "nonbonded_potential_tilegroups.cui"
#          undef KERNEL_NAME
#        undef DO_NECK_CORRECTION
#      undef DO_GENERALIZED_BORN
#      undef NONBOND_KERNEL_BLOCKS_MULTIPLIER
#    undef COMPUTE_ENERGY
#  undef CLASH_FORGIVENESS
#  undef LLCONV_FUNC
#  undef SQRT_FUNC
#  undef CBRT_FUNC
#  undef LOG_FUNC
#  undef EXP_FUNC
#  undef TANH_FUNC
#  undef FABS_FUNC
#  undef TCALC_IS_SINGLE
#  undef TCALC2
#undef TCALC

// Double-precision non-bonded kernel floating point definitions
#define TCALC double
#  define TCALC2 double2
#  define SPLIT_FORCE_ACCUMULATION
#  define NONBOND_KERNEL_BLOCKS_MULTIPLIER 3
#  define GBRADII_KERNEL_BLOCKS_MULTIPLIER 3
#  define GBDERIV_KERNEL_BLOCKS_MULTIPLIER 3
#  define LLCONV_FUNC __double2ll_rn
#  define SQRT_FUNC sqrt
#  define CBRT_FUNC cbrt
#  define LOG_FUNC  log
#  define EXP_FUNC  exp
#  define TANH_FUNC tanh
#  define FABS_FUNC fabs
#  define KERNEL_NAME ktgdsCalculateGBRadii
#    include "gbradii_tilegroups.cui"
#  undef KERNEL_NAME
#  define DO_NECK_CORRECTION
#    define KERNEL_NAME ktgdsCalculateGBNeckRadii
#      include "gbradii_tilegroups.cui"
#    undef KERNEL_NAME
#  undef DO_NECK_CORRECTION
#  define COMPUTE_FORCE
#    define COMPUTE_ENERGY
#      define KERNEL_NAME ktgdsVacuumForceEnergy
#        include "nonbonded_potential_tilegroups.cui"
#      undef KERNEL_NAME
#      define DO_GENERALIZED_BORN
#        define KERNEL_NAME ktgdsGBForceEnergy
#          include "nonbonded_potential_tilegroups.cui"
#        undef KERNEL_NAME
#        define KERNEL_NAME ktgdsCalculateGBDerivatives
#          include "gbderivative_tilegroups.cui"
#        undef KERNEL_NAME
#        define DO_NECK_CORRECTION
#          define KERNEL_NAME ktgdsGBNeckForceEnergy
#            include "nonbonded_potential_tilegroups.cui"
#          undef KERNEL_NAME
#          define KERNEL_NAME ktgdsCalculateGBNeckDerivatives
#            include "gbderivative_tilegroups.cui"
#          undef KERNEL_NAME
#        undef DO_NECK_CORRECTION
#      undef DO_GENERALIZED_BORN
#    undef COMPUTE_ENERGY
#    define KERNEL_NAME ktgdsVacuumForce
#      include "nonbonded_potential_tilegroups.cui"
#    undef KERNEL_NAME
#    define DO_GENERALIZED_BORN
#      define KERNEL_NAME ktgdsGBForce
#        include "nonbonded_potential_tilegroups.cui"
#      undef KERNEL_NAME
#      define DO_NECK_CORRECTION
#        define KERNEL_NAME ktgdsGBNeckForce
#          include "nonbonded_potential_tilegroups.cui"
#        undef KERNEL_NAME
#      undef DO_NECK_CORRECTION
#    undef DO_GENERALIZED_BORN
#  undef COMPUTE_FORCE
#  define COMPUTE_ENERGY
#    define KERNEL_NAME ktgdVacuumEnergy
#      include "nonbonded_potential_tilegroups.cui"
#    undef KERNEL_NAME
#    define DO_GENERALIZED_BORN
#      define KERNEL_NAME ktgdGBEnergy
#        include "nonbonded_potential_tilegroups.cui"
#      undef KERNEL_NAME
#      define DO_NECK_CORRECTION
#        define KERNEL_NAME ktgdGBNeckEnergy
#          include "nonbonded_potential_tilegroups.cui"
#        undef KERNEL_NAME
#      undef DO_NECK_CORRECTION
#    undef DO_GENERALIZED_BORN
#  undef COMPUTE_ENERGY
#  undef GBRADII_KERNEL_BLOCKS_MULTIPLIER
#  undef GBDERIV_KERNEL_BLOCKS_MULTIPLIER
#  undef NONBOND_KERNEL_BLOCKS_MULTIPLIER
#  define CLASH_FORGIVENESS
#    define COMPUTE_FORCE
#      define COMPUTE_ENERGY
#        define NONBOND_KERNEL_BLOCKS_MULTIPLIER 3
#        define KERNEL_NAME ktgdsVacuumForceEnergyNonClash
#          include "nonbonded_potential_tilegroups.cui"
#        undef KERNEL_NAME
#        undef NONBOND_KERNEL_BLOCKS_MULTIPLIER
#        define DO_GENERALIZED_BORN
#          define NONBOND_KERNEL_BLOCKS_MULTIPLIER 2
#          define KERNEL_NAME ktgdsGBForceEnergyNonClash
#            include "nonbonded_potential_tilegroups.cui"
#          undef KERNEL_NAME
#          define DO_NECK_CORRECTION
#            define KERNEL_NAME ktgdsGBNeckForceEnergyNonClash
#              include "nonbonded_potential_tilegroups.cui"
#            undef KERNEL_NAME
#          undef DO_NECK_CORRECTION
#          undef NONBOND_KERNEL_BLOCKS_MULTIPLIER
#        undef DO_GENERALIZED_BORN
#      undef COMPUTE_ENERGY
#      define NONBOND_KERNEL_BLOCKS_MULTIPLIER 3
#      define KERNEL_NAME ktgdsVacuumForceNonClash
#        include "nonbonded_potential_tilegroups.cui"
#      undef KERNEL_NAME
#      define DO_GENERALIZED_BORN
#        define KERNEL_NAME ktgdsGBForceNonClash
#          include "nonbonded_potential_tilegroups.cui"
#        undef KERNEL_NAME
#        define DO_NECK_CORRECTION
#          define KERNEL_NAME ktgdsGBNeckForceNonClash
#            include "nonbonded_potential_tilegroups.cui"
#          undef KERNEL_NAME
#        undef DO_NECK_CORRECTION
#      undef DO_GENERALIZED_BORN
#      undef NONBOND_KERNEL_BLOCKS_MULTIPLIER
#    undef COMPUTE_FORCE
#    define COMPUTE_ENERGY
#      define NONBOND_KERNEL_BLOCKS_MULTIPLIER 3
#      define KERNEL_NAME ktgdVacuumEnergyNonClash
#        include "nonbonded_potential_tilegroups.cui"
#      undef KERNEL_NAME
#      define DO_GENERALIZED_BORN
#        define KERNEL_NAME ktgdGBEnergyNonClash
#          include "nonbonded_potential_tilegroups.cui"
#        undef KERNEL_NAME
#        define DO_NECK_CORRECTION
#          define KERNEL_NAME ktgdGBNeckEnergyNonClash
#            include "nonbonded_potential_tilegroups.cui"
#          undef KERNEL_NAME
#        undef DO_NECK_CORRECTION
#      undef DO_GENERALIZED_BORN
#      undef NONBOND_KERNEL_BLOCKS_MULTIPLIER
#    undef COMPUTE_ENERGY
#  undef CLASH_FORGIVENESS
#  undef LLCONV_FUNC
#  undef SQRT_FUNC
#  undef CBRT_FUNC
#  undef LOG_FUNC
#  undef EXP_FUNC
#  undef TANH_FUNC
#  undef FABS_FUNC
#  undef SPLIT_FORCE_ACCUMULATION
#  undef TCALC2
#undef TCALC

//-------------------------------------------------------------------------------------------------
extern hipFuncAttributes
queryNonbondedKernelRequirements(const PrecisionModel prec, const NbwuKind kind,
                                 const EvaluateForce eval_frc, const EvaluateEnergy eval_nrg,
                                 const AccumulationMethod acc_meth,
                                 const ImplicitSolventModel igb,
                                 const ClashResponse collision_handling) {
  
  // The kernel manager will have information about the GPU to use--look at the work units from
  // the perspective of overall occupancy on the GPU.
  hipFuncAttributes attr;
  switch (collision_handling) {
  case ClashResponse::NONE:
    switch (igb) {
    case ImplicitSolventModel::NONE:
      switch (prec) {
      case PrecisionModel::DOUBLE:
        switch (eval_frc) {
        case EvaluateForce::YES:
          switch (eval_nrg) {
          case EvaluateEnergy::YES:
            if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgdsVacuumForceEnergy)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel ktgdsVacuumForceEnergy.",
                    "queryNonbondedKernelRequirements");
            }
            break;
          case EvaluateEnergy::NO:
            if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgdsVacuumForce)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel ktgdsVacuumForce.",
                    "queryNonbondedKernelRequirements");
            }
            break;
          }
          break;
        case EvaluateForce::NO:
          if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgdVacuumEnergy)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel ktgdVacuumEnergy.",
                  "queryNonbondedKernelRequirements");
          }
          break;
        }
        break;
      case PrecisionModel::SINGLE:
        switch (kind) {
          case NbwuKind::TILE_GROUPS:
          switch (eval_frc) {
          case EvaluateForce::YES:
            switch (eval_nrg) {
            case EvaluateEnergy::YES:
              switch (acc_meth) {
              case AccumulationMethod::SPLIT:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfsVacuumForceEnergy)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfsVacuumForceEnergy.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              case AccumulationMethod::WHOLE:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfVacuumForceEnergy)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfVacuumForceEnergy.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              }
              break;
            case EvaluateEnergy::NO:
              switch (acc_meth) {
              case AccumulationMethod::SPLIT:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfsVacuumForce)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfsVacuumForce.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              case AccumulationMethod::WHOLE:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfVacuumForce)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfVacuumForce.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              }
              break;
            }
            break;
          case EvaluateForce::NO:
            if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfVacuumEnergy)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel ktgfVacuumEnergy.",
                    "queryNonbondedKernelRequirements");
            }
            break;
          }
          break;
        case NbwuKind::SUPERTILES:
          break;
        case NbwuKind::HONEYCOMB:
          break;
        }
        break;
      }
      break;
    case ImplicitSolventModel::HCT_GB:
    case ImplicitSolventModel::OBC_GB:
    case ImplicitSolventModel::OBC_GB_II:
      switch (prec) {
      case PrecisionModel::DOUBLE:
        switch (eval_frc) {
        case EvaluateForce::YES:
          switch (eval_nrg) {
          case EvaluateEnergy::YES:
            if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgdsGBForceEnergy)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel ktgdsGBForceEnergy.",
                    "queryNonbondedKernelRequirements");
            }
            break;
          case EvaluateEnergy::NO:
            if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgdsGBForce)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel ktgdsGBForce.",
                    "queryNonbondedKernelRequirements");
            }
            break;
          }
          break;
        case EvaluateForce::NO:
          if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgdGBEnergy)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel ktgdGBEnergy.",
                  "queryNonbondedKernelRequirements");
          }
          break;
        }
        break;
      case PrecisionModel::SINGLE:
        switch (kind) {
          case NbwuKind::TILE_GROUPS:
          switch (eval_frc) {
          case EvaluateForce::YES:
            switch (eval_nrg) {
            case EvaluateEnergy::YES:
              switch (acc_meth) {
              case AccumulationMethod::SPLIT:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfsGBForceEnergy)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfsGBForceEnergy.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              case AccumulationMethod::WHOLE:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfGBForceEnergy)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfGBForceEnergy.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              }
              break;
            case EvaluateEnergy::NO:
              switch (acc_meth) {
              case AccumulationMethod::SPLIT:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfsGBForce)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfsGBForce.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              case AccumulationMethod::WHOLE:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfGBForce)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfGBForce.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              }
              break;
            }
            break;
          case EvaluateForce::NO:
            if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfGBEnergy)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel ktgfGBEnergy.",
                    "queryNonbondedKernelRequirements");
            }
            break;
          }
          break;
        case NbwuKind::SUPERTILES:
          break;
        case NbwuKind::HONEYCOMB:
          break;
        }
        break;
      }
      break;
    case ImplicitSolventModel::NECK_GB:
    case ImplicitSolventModel::NECK_GB_II:
      switch (prec) {
      case PrecisionModel::DOUBLE:
        switch (eval_frc) {
        case EvaluateForce::YES:
          switch (eval_nrg) {
          case EvaluateEnergy::YES:
            if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgdsGBNeckForceEnergy)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel ktgdsGBNeckForceEnergy.",
                    "queryNonbondedKernelRequirements");
            }
            break;
          case EvaluateEnergy::NO:
            if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgdsGBNeckForce)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel ktgdsGBNeckForce.",
                    "queryNonbondedKernelRequirements");
            }
            break;
          }
          break;
        case EvaluateForce::NO:
          if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgdGBNeckEnergy)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel ktgdGBNeckEnergy.",
                  "queryNonbondedKernelRequirements");
          }
          break;
        }
        break;
      case PrecisionModel::SINGLE:
        switch (kind) {
          case NbwuKind::TILE_GROUPS:
          switch (eval_frc) {
          case EvaluateForce::YES:
            switch (eval_nrg) {
            case EvaluateEnergy::YES:
              switch (acc_meth) {
              case AccumulationMethod::SPLIT:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfsGBNeckForceEnergy)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfsGBNeckForceEnergy.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              case AccumulationMethod::WHOLE:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfGBNeckForceEnergy)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfGBNeckForceEnergy.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              }
              break;
            case EvaluateEnergy::NO:
              switch (acc_meth) {
              case AccumulationMethod::SPLIT:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfsGBNeckForce)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfsGBNeckForce.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              case AccumulationMethod::WHOLE:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfGBNeckForce)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfGBNeckForce.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              }
              break;
            }
            break;
          case EvaluateForce::NO:
            if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfGBNeckEnergy)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel ktgfGBNeckEnergy.",
                    "queryNonbondedKernelRequirements");
            }
            break;
          }
          break;
        case NbwuKind::SUPERTILES:
          break;
        case NbwuKind::HONEYCOMB:
          break;
        }
        break;
      }
      break;
    }
    break;
  case ClashResponse::FORGIVE:
    switch (igb) {
    case ImplicitSolventModel::NONE:
      switch (prec) {
      case PrecisionModel::DOUBLE:
        switch (eval_frc) {
        case EvaluateForce::YES:
          switch (eval_nrg) {
          case EvaluateEnergy::YES:
            if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgdsVacuumForceEnergyNonClash)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel ktgdsVacuumForceEnergyNonClash.",
                    "queryNonbondedKernelRequirements");
            }
            break;
          case EvaluateEnergy::NO:
            if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgdsVacuumForceNonClash)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel ktgdsVacuumForceNonClash.",
                    "queryNonbondedKernelRequirements");
            }
            break;
          }
          break;
        case EvaluateForce::NO:
          if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgdVacuumEnergyNonClash)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel ktgdVacuumEnergyNonClash.",
                  "queryNonbondedKernelRequirements");
          }
          break;
        }
        break;
      case PrecisionModel::SINGLE:
        switch (kind) {
          case NbwuKind::TILE_GROUPS:
          switch (eval_frc) {
          case EvaluateForce::YES:
            switch (eval_nrg) {
            case EvaluateEnergy::YES:
              switch (acc_meth) {
              case AccumulationMethod::SPLIT:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfsVacuumForceEnergyNonClash)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfsVacuumForceEnergyNonClash.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              case AccumulationMethod::WHOLE:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfVacuumForceEnergyNonClash)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfVacuumForceEnergyNonClash.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              }
              break;
            case EvaluateEnergy::NO:
              switch (acc_meth) {
              case AccumulationMethod::SPLIT:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfsVacuumForceNonClash)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfsVacuumForceNonClash.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              case AccumulationMethod::WHOLE:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfVacuumForceNonClash)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfVacuumForceNonClash.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              }
              break;
            }
            break;
          case EvaluateForce::NO:
            if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfVacuumEnergyNonClash)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel ktgfVacuumEnergyNonClash.",
                    "queryNonbondedKernelRequirements");
            }
            break;
          }
          break;
        case NbwuKind::SUPERTILES:
          break;
        case NbwuKind::HONEYCOMB:
          break;
        }
        break;
      }
      break;
    case ImplicitSolventModel::HCT_GB:
    case ImplicitSolventModel::OBC_GB:
    case ImplicitSolventModel::OBC_GB_II:
      switch (prec) {
      case PrecisionModel::DOUBLE:
        switch (eval_frc) {
        case EvaluateForce::YES:
          switch (eval_nrg) {
          case EvaluateEnergy::YES:
            if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgdsGBForceEnergyNonClash)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel ktgdsGBForceEnergyNonClash.",
                    "queryNonbondedKernelRequirements");
            }
            break;
          case EvaluateEnergy::NO:
            if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgdsGBForceNonClash)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel ktgdsGBForceNonClash.",
                    "queryNonbondedKernelRequirements");
            }
            break;
          }
          break;
        case EvaluateForce::NO:
          if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgdGBEnergyNonClash)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel ktgdGBEnergyNonClash.",
                  "queryNonbondedKernelRequirements");
          }
          break;
        }
        break;
      case PrecisionModel::SINGLE:
        switch (kind) {
          case NbwuKind::TILE_GROUPS:
          switch (eval_frc) {
          case EvaluateForce::YES:
            switch (eval_nrg) {
            case EvaluateEnergy::YES:
              switch (acc_meth) {
              case AccumulationMethod::SPLIT:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfsGBForceEnergyNonClash)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfsGBForceEnergyNonClash.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              case AccumulationMethod::WHOLE:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfGBForceEnergyNonClash)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfGBForceEnergyNonClash.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              }
              break;
            case EvaluateEnergy::NO:
              switch (acc_meth) {
              case AccumulationMethod::SPLIT:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfsGBForceNonClash)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfsGBForceNonClash.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              case AccumulationMethod::WHOLE:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfGBForceNonClash)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfGBForceNonClash.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              }
              break;
            }
            break;
          case EvaluateForce::NO:
            if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfGBEnergyNonClash)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel ktgfGBEnergyNonClash.",
                    "queryNonbondedKernelRequirements");
            }
            break;
          }
          break;
        case NbwuKind::SUPERTILES:
          break;
        case NbwuKind::HONEYCOMB:
          break;
        }
        break;
      }
      break;
    case ImplicitSolventModel::NECK_GB:
    case ImplicitSolventModel::NECK_GB_II:
      switch (prec) {
      case PrecisionModel::DOUBLE:
        switch (eval_frc) {
        case EvaluateForce::YES:
          switch (eval_nrg) {
          case EvaluateEnergy::YES:
            if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgdsGBNeckForceEnergyNonClash)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel ktgdsGBNeckForceEnergyNonClash.",
                    "queryNonbondedKernelRequirements");
            }
            break;
          case EvaluateEnergy::NO:
            if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgdsGBNeckForceNonClash)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel ktgdsGBNeckForceNonClash.",
                    "queryNonbondedKernelRequirements");
            }
            break;
          }
          break;
        case EvaluateForce::NO:
          if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgdGBNeckEnergyNonClash)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel ktgdGBNeckEnergyNonClash.",
                  "queryNonbondedKernelRequirements");
          }
          break;
        }
        break;
      case PrecisionModel::SINGLE:
        switch (kind) {
          case NbwuKind::TILE_GROUPS:
          switch (eval_frc) {
          case EvaluateForce::YES:
            switch (eval_nrg) {
            case EvaluateEnergy::YES:
              switch (acc_meth) {
              case AccumulationMethod::SPLIT:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfsGBNeckForceEnergyNonClash)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfsGBNeckForceEnergyNonClash.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              case AccumulationMethod::WHOLE:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfGBNeckForceEnergyNonClash)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfGBNeckForceEnergyNonClash.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              }
              break;
            case EvaluateEnergy::NO:
              switch (acc_meth) {
              case AccumulationMethod::SPLIT:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfsGBNeckForceNonClash)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfsGBNeckForceNonClash.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              case AccumulationMethod::WHOLE:
                if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfGBNeckForceNonClash)) != hipSuccess) {
                  rtErr("Error obtaining attributes for kernel ktgfGBNeckForceNonClash.",
                        "queryNonbondedKernelRequirements");
                }
                break;
              }
              break;
            }
            break;
          case EvaluateForce::NO:
            if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfGBNeckEnergyNonClash)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel ktgfGBNeckEnergyNonClash.",
                    "queryNonbondedKernelRequirements");
            }
            break;
          }
          break;
        case NbwuKind::SUPERTILES:
          break;
        case NbwuKind::HONEYCOMB:
          break;
        }
        break;
      }
      break;
    }
    break;
  }
  return attr;
}

//-------------------------------------------------------------------------------------------------
extern hipFuncAttributes
queryBornRadiiKernelRequirements(const PrecisionModel prec, const NbwuKind kind,
                                 const AccumulationMethod acc_meth,
                                 const ImplicitSolventModel igb) {
  hipFuncAttributes attr;
  switch (igb) {
  case ImplicitSolventModel::NONE:
    break;
  case ImplicitSolventModel::HCT_GB:
  case ImplicitSolventModel::OBC_GB:
  case ImplicitSolventModel::OBC_GB_II:
    switch (prec) {
    case PrecisionModel::DOUBLE:
      switch (kind) {
      case NbwuKind::TILE_GROUPS:
        if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgdsCalculateGBRadii)) != hipSuccess) {
          rtErr("Error obtaining attributes for kernel ktgdCalculateGBRadii.",
                "queryNonbondedKernelRequirements");
        }
        break;
      case NbwuKind::SUPERTILES:
      case NbwuKind::HONEYCOMB:
        break;
      }
      break;
    case PrecisionModel::SINGLE:
      switch (kind) {
      case NbwuKind::TILE_GROUPS:
        switch (acc_meth) {
        case AccumulationMethod::SPLIT:
          if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfsCalculateGBRadii)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel ktgfsCalculateGBRadii.",
                  "queryNonbondedKernelRequirements");
          }
          break;
        case AccumulationMethod::WHOLE:
          if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfCalculateGBRadii)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel ktgfCalculateGBRadii.",
                  "queryNonbondedKernelRequirements");
          }
          break;
        case AccumulationMethod::AUTOMATIC:
          break;
        }
        break;
      case NbwuKind::SUPERTILES:
      case NbwuKind::HONEYCOMB:
        break;
      }
      break;
    }
    break;
  case ImplicitSolventModel::NECK_GB:
  case ImplicitSolventModel::NECK_GB_II:
    switch (prec) {
    case PrecisionModel::DOUBLE:
      switch (kind) {
      case NbwuKind::TILE_GROUPS:
        if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgdsCalculateGBNeckRadii)) != hipSuccess) {
          rtErr("Error obtaining attributes for kernel ktgdCalculateGBNeckRadii.",
                "queryNonbondedKernelRequirements");
        }
        break;
      case NbwuKind::SUPERTILES:
      case NbwuKind::HONEYCOMB:
        break;
      }
      break;
    case PrecisionModel::SINGLE:
      switch (kind) {
      case NbwuKind::TILE_GROUPS:
        switch (acc_meth) {
        case AccumulationMethod::SPLIT:
          if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfsCalculateGBNeckRadii)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel ktgfsCalculateGBNeckRadii.",
                  "queryNonbondedKernelRequirements");
          }
          break;
        case AccumulationMethod::WHOLE:
          if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfCalculateGBNeckRadii)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel ktgfCalculateGBNeckRadii.",
                  "queryNonbondedKernelRequirements");
          }
          break;
        case AccumulationMethod::AUTOMATIC:
          break;
        }
        break;
      case NbwuKind::SUPERTILES:
      case NbwuKind::HONEYCOMB:
        break;
      }
      break;
    }
    break;
  }
  return attr;
}

//-------------------------------------------------------------------------------------------------
extern hipFuncAttributes
queryBornDerivativeKernelRequirements(const PrecisionModel prec, const NbwuKind kind,
                                      const AccumulationMethod acc_meth,
                                      const ImplicitSolventModel igb) {
  hipFuncAttributes attr;
  switch (igb) {
  case ImplicitSolventModel::NONE:
    break;
  case ImplicitSolventModel::HCT_GB:
  case ImplicitSolventModel::OBC_GB:
  case ImplicitSolventModel::OBC_GB_II:
    switch (prec) {
    case PrecisionModel::DOUBLE:
      switch (kind) {
      case NbwuKind::TILE_GROUPS:
        if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgdsCalculateGBDerivatives)) != hipSuccess) {
          rtErr("Error obtaining attributes for kernel ktgdCalculateGBDerivatives.",
                "queryNonbondedKernelRequirements");
        }
        break;
      case NbwuKind::SUPERTILES:
      case NbwuKind::HONEYCOMB:
        break;
      }
      break;
    case PrecisionModel::SINGLE:
      switch (kind) {
      case NbwuKind::TILE_GROUPS:
        switch (acc_meth) {
        case AccumulationMethod::SPLIT:
          if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfsCalculateGBDerivatives)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel ktgfsCalculateGBDerivatives.",
                  "queryNonbondedKernelRequirements");
          }
          break;
        case AccumulationMethod::WHOLE:
          if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfCalculateGBDerivatives)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel ktgfCalculateGBDerivatives.",
                  "queryNonbondedKernelRequirements");
          }
          break;
        case AccumulationMethod::AUTOMATIC:
          break;
        }
        break;
      case NbwuKind::SUPERTILES:
      case NbwuKind::HONEYCOMB:
        break;
      }
      break;
    }
    break;
  case ImplicitSolventModel::NECK_GB:
  case ImplicitSolventModel::NECK_GB_II:
    switch (prec) {
    case PrecisionModel::DOUBLE:
      switch (kind) {
      case NbwuKind::TILE_GROUPS:
        if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgdsCalculateGBNeckDerivatives)) != hipSuccess) {
          rtErr("Error obtaining attributes for kernel ktgdCalculateGBNeckDerivatives.",
                "queryNonbondedKernelRequirements");
        }
        break;
      case NbwuKind::SUPERTILES:
      case NbwuKind::HONEYCOMB:
        break;
      }
      break;
    case PrecisionModel::SINGLE:
      switch (kind) {
      case NbwuKind::TILE_GROUPS:
        switch (acc_meth) {
        case AccumulationMethod::SPLIT:
          if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfsCalculateGBNeckDerivatives)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel ktgfsCalculateGBNeckDerivatives.",
                  "queryNonbondedKernelRequirements");
          }
          break;
        case AccumulationMethod::WHOLE:
          if (hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(ktgfCalculateGBNeckDerivatives)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel ktgfCalculateGBNeckDerivatives.",
                  "queryNonbondedKernelRequirements");
          }
          break;
        case AccumulationMethod::AUTOMATIC:
          break;
        }
        break;
      case NbwuKind::SUPERTILES:
      case NbwuKind::HONEYCOMB:
        break;
      }
      break;
    }
    break;
  }    
  return attr;
}

//-------------------------------------------------------------------------------------------------
extern void launchNonbonded(const NbwuKind kind, const SyNonbondedKit<double, double2> &poly_nbk,
                            const SeMaskSynthesisReader &poly_ser, MMControlKit<double> *ctrl,
                            PsSynthesisWriter *poly_psw, ThermostatWriter<double> *tstw,
                            ScoreCardWriter *scw, CacheResourceKit<double> *gmem_r,
                            ISWorkspaceKit<double> *iswk, const EvaluateForce eval_force,
                            const EvaluateEnergy eval_energy, const int2 bt, const int2 gbr_bt,
                            const int2 gbd_bt, const double clash_minimum_distance,
                            const double clash_ratio) {

  // As with the valence kernels, detect the intention to intercept and dampen clashes by nonzero
  // values in the parameters, not an explicit input based on the ClashResponse enumerator.
  if (clash_minimum_distance >= 1.0e-6 || clash_ratio >= 1.0e-6) {
    switch (kind) {
    case NbwuKind::TILE_GROUPS:
      switch (poly_nbk.igb) {
      case ImplicitSolventModel::NONE:
        switch (eval_force) {
        case EvaluateForce::YES:
          switch (eval_energy) {
          case EvaluateEnergy::YES:
            ktgdsVacuumForceEnergyNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                                           clash_minimum_distance, clash_ratio,
                                                           *scw, *tstw, *gmem_r);
            break;
          case EvaluateEnergy::NO:
            ktgdsVacuumForceNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                                     clash_minimum_distance, clash_ratio, *tstw,
                                                     *gmem_r);
            break;
          }
          break;
        case EvaluateForce::NO:
          ktgdVacuumEnergyNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                                   clash_minimum_distance, clash_ratio, *scw,
                                                   *tstw, *gmem_r);
          break;
        }
        break;
      case ImplicitSolventModel::HCT_GB:
      case ImplicitSolventModel::OBC_GB:
      case ImplicitSolventModel::OBC_GB_II:
        ktgdsCalculateGBRadii<<<gbr_bt.x, gbr_bt.y>>>(poly_nbk, *ctrl, *poly_psw, *tstw, *iswk,
                                                      *gmem_r);
        switch (eval_force) {
        case EvaluateForce::YES:
          switch (eval_energy) {
          case EvaluateEnergy::YES:
            ktgdsGBForceEnergyNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                                       clash_minimum_distance, clash_ratio, *scw,
                                                       *iswk, *gmem_r);
            break;
          case EvaluateEnergy::NO:
            ktgdsGBForceNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                                 clash_minimum_distance, clash_ratio, *iswk,
                                                 *gmem_r);
            break;
          }
          ktgdsCalculateGBDerivatives<<<gbd_bt.x, gbd_bt.y>>>(poly_nbk, *ctrl, *poly_psw, *iswk,
                                                              *gmem_r);
          break;
        case EvaluateForce::NO:
          ktgdGBEnergyNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                               clash_minimum_distance, clash_ratio, *scw, *iswk,
                                               *gmem_r);
          break;
        }
        break;
      case ImplicitSolventModel::NECK_GB:
      case ImplicitSolventModel::NECK_GB_II:
        ktgdsCalculateGBNeckRadii<<<gbr_bt.x, gbr_bt.y>>>(poly_nbk, *ctrl, *poly_psw, *tstw, *iswk,
                                                          *gmem_r);
        switch (eval_force) {
        case EvaluateForce::YES:
          switch (eval_energy) {
          case EvaluateEnergy::YES:
            ktgdsGBNeckForceEnergyNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                                           clash_minimum_distance, clash_ratio,
                                                           *scw, *iswk, *gmem_r);
            break;
          case EvaluateEnergy::NO:
            ktgdsGBNeckForceNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                                     clash_minimum_distance, clash_ratio, *iswk,
                                                     *gmem_r);
            break;
          }
          ktgdsCalculateGBNeckDerivatives<<<gbd_bt.x, gbd_bt.y>>>(poly_nbk, *ctrl, *poly_psw,
                                                                  *iswk, *gmem_r);
          break;
        case EvaluateForce::NO:
          ktgdGBNeckEnergyNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                                   clash_minimum_distance, clash_ratio, *scw,
                                                   *iswk, *gmem_r);
          break;
        }
        break;
      }
      break;
    case NbwuKind::SUPERTILES:
    case NbwuKind::HONEYCOMB:
      break;
    }
  }
  else {
    switch (kind) {
    case NbwuKind::TILE_GROUPS:
      switch (poly_nbk.igb) {
      case ImplicitSolventModel::NONE:
        switch (eval_force) {
        case EvaluateForce::YES:
          switch (eval_energy) {
          case EvaluateEnergy::YES:
            ktgdsVacuumForceEnergy<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *scw,
                                                   *tstw, *gmem_r);
            break;
          case EvaluateEnergy::NO:
            ktgdsVacuumForce<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *tstw, *gmem_r);
            break;
          }
          break;
        case EvaluateForce::NO:
          ktgdVacuumEnergy<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *scw, *tstw,
                                           *gmem_r);
          break;
        }
        break;
      case ImplicitSolventModel::HCT_GB:
      case ImplicitSolventModel::OBC_GB:
      case ImplicitSolventModel::OBC_GB_II:
        ktgdsCalculateGBRadii<<<gbr_bt.x, gbr_bt.y>>>(poly_nbk, *ctrl, *poly_psw, *tstw, *iswk,
                                                      *gmem_r);
        switch (eval_force) {
        case EvaluateForce::YES:
          switch (eval_energy) {
          case EvaluateEnergy::YES:
            ktgdsGBForceEnergy<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *scw,
                                               *iswk, *gmem_r);
            break;
          case EvaluateEnergy::NO:
            ktgdsGBForce<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *iswk,
                                         *gmem_r);
            break;
          }
          ktgdsCalculateGBDerivatives<<<gbd_bt.x, gbd_bt.y>>>(poly_nbk, *ctrl, *poly_psw, *iswk,
                                                              *gmem_r);
          break;
        case EvaluateForce::NO:
          ktgdGBEnergy<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *scw, *iswk, *gmem_r);
          break;
        }
        break;
      case ImplicitSolventModel::NECK_GB:
      case ImplicitSolventModel::NECK_GB_II:
        ktgdsCalculateGBNeckRadii<<<gbr_bt.x, gbr_bt.y>>>(poly_nbk, *ctrl, *poly_psw, *tstw, *iswk,
                                                          *gmem_r);
        switch (eval_force) {
        case EvaluateForce::YES:
          switch (eval_energy) {
          case EvaluateEnergy::YES:
            ktgdsGBNeckForceEnergy<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *scw,
                                                   *iswk, *gmem_r);
            break;
          case EvaluateEnergy::NO:
            ktgdsGBNeckForce<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *iswk, *gmem_r);
            break;
          }
          ktgdsCalculateGBNeckDerivatives<<<gbd_bt.x, gbd_bt.y>>>(poly_nbk, *ctrl, *poly_psw,
                                                                  *iswk, *gmem_r);
          break;
        case EvaluateForce::NO:
          ktgdGBNeckEnergy<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *scw, *iswk,
                                           *gmem_r);
          break;
        }
        break;
      }
      break;
    case NbwuKind::SUPERTILES:
    case NbwuKind::HONEYCOMB:
      break;
    }
  }
}

//-------------------------------------------------------------------------------------------------
extern void launchNonbonded(const NbwuKind kind, const SyNonbondedKit<float, float2> &poly_nbk,
                            const SeMaskSynthesisReader &poly_ser, MMControlKit<float> *ctrl,
                            PsSynthesisWriter *poly_psw, ThermostatWriter<float> *tstw,
                            ScoreCardWriter *scw, CacheResourceKit<float> *gmem_r,
                            ISWorkspaceKit<float> *iswk, const EvaluateForce eval_force,
                            const EvaluateEnergy eval_energy, const AccumulationMethod force_sum,
                            const int2 bt, const int2 gbr_bt, const int2 gbd_bt,
                            const float clash_minimum_distance, const float clash_ratio) {
  const AccumulationMethod actual_force_sum = (force_sum == AccumulationMethod::AUTOMATIC) ?
                                              chooseAccumulationMethod(poly_psw->frc_bits) :
                                              force_sum;
  // As with the valence kernels, detect the intention to intercept and dampen clashes by nonzero
  // values in the parameters, not an explicit input based on the ClashResponse enumerator.
  if (clash_minimum_distance > 0.000001f || clash_ratio > 0.000001f) {
    switch (kind) {
    case NbwuKind::TILE_GROUPS:
      switch (poly_nbk.igb) {
      case ImplicitSolventModel::NONE:
        switch (eval_force) {
        case EvaluateForce::YES:
          switch (actual_force_sum) {
          case AccumulationMethod::SPLIT:
            switch (eval_energy) {
            case EvaluateEnergy::YES:
              ktgfsVacuumForceEnergyNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                                             clash_minimum_distance, clash_ratio,
                                                             *scw, *tstw, *gmem_r);
              break;
            case EvaluateEnergy::NO:
              ktgfsVacuumForceNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                                       clash_minimum_distance, clash_ratio, *tstw,
                                                       *gmem_r);
              break;
            }
            break;
          case AccumulationMethod::WHOLE:
            switch (eval_energy) {
            case EvaluateEnergy::YES:
              ktgfVacuumForceEnergyNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                                            clash_minimum_distance, clash_ratio,
                                                            *scw, *tstw, *gmem_r);
              break;
            case EvaluateEnergy::NO:
              ktgfVacuumForceNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                                      clash_minimum_distance, clash_ratio, *tstw,
                                                      *gmem_r);
              break;
            }
            break;
          case AccumulationMethod::AUTOMATIC:

            // This case was converted into SPLIT or WHOLE by evaluating actual_force_sum
            break;
          }
          break;
        case EvaluateForce::NO:
          ktgfVacuumEnergyNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                                   clash_minimum_distance, clash_ratio, *scw,
                                                   *tstw, *gmem_r);
          break;
        }
        break;
      case ImplicitSolventModel::HCT_GB:
      case ImplicitSolventModel::OBC_GB:
      case ImplicitSolventModel::OBC_GB_II:
        switch (actual_force_sum) {
        case AccumulationMethod::SPLIT:
          ktgfsCalculateGBRadii<<<gbr_bt.x, gbr_bt.y>>>(poly_nbk, *ctrl, *poly_psw, *tstw, *iswk,
                                                        *gmem_r);
          break;
        case AccumulationMethod::WHOLE:
          ktgfCalculateGBRadii<<<gbr_bt.x, gbr_bt.y>>>(poly_nbk, *ctrl, *poly_psw, *tstw, *iswk,
                                                       *gmem_r);
          break;
        case AccumulationMethod::AUTOMATIC:
          break;
        }
        switch (eval_force) {
        case EvaluateForce::YES:
          switch (actual_force_sum) {
          case AccumulationMethod::SPLIT:
            switch (eval_energy) {
            case EvaluateEnergy::YES:
              ktgfsGBForceEnergyNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                                         clash_minimum_distance, clash_ratio, *scw,
                                                         *iswk, *gmem_r);
              break;
            case EvaluateEnergy::NO:
              ktgfsGBForceNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                                   clash_minimum_distance, clash_ratio, *iswk,
                                                   *gmem_r);
              break;
            }
            ktgfsCalculateGBDerivatives<<<gbd_bt.x, gbd_bt.y>>>(poly_nbk, *ctrl, *poly_psw, *iswk,
                                                                *gmem_r);
            break;
          case AccumulationMethod::WHOLE:
            switch (eval_energy) {
            case EvaluateEnergy::YES:
              ktgfGBForceEnergyNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                                        clash_minimum_distance, clash_ratio, *scw,
                                                        *iswk, *gmem_r);
              break;
            case EvaluateEnergy::NO:
              ktgfGBForceNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                                  clash_minimum_distance, clash_ratio, *iswk,
                                                  *gmem_r);
              break;
            }
            ktgfCalculateGBDerivatives<<<gbd_bt.x, gbd_bt.y>>>(poly_nbk, *ctrl, *poly_psw, *iswk,
                                                               *gmem_r);
            break;
          case AccumulationMethod::AUTOMATIC:
            break;
          }
          break;
        case EvaluateForce::NO:
          ktgfGBEnergyNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                               clash_minimum_distance, clash_ratio, *scw, *iswk,
                                               *gmem_r);
          break;
        }
        break;
      case ImplicitSolventModel::NECK_GB:
      case ImplicitSolventModel::NECK_GB_II:
        switch (actual_force_sum) {
        case AccumulationMethod::SPLIT:
          ktgfsCalculateGBNeckRadii<<<gbr_bt.x, gbr_bt.y>>>(poly_nbk, *ctrl, *poly_psw, *tstw,
                                                            *iswk, *gmem_r);
          break;
        case AccumulationMethod::WHOLE:
          ktgfCalculateGBNeckRadii<<<gbr_bt.x, gbr_bt.y>>>(poly_nbk, *ctrl, *poly_psw, *tstw,
                                                           *iswk, *gmem_r);
          break;
        case AccumulationMethod::AUTOMATIC:
          break;
        }
        switch (eval_force) {
        case EvaluateForce::YES:
          switch (actual_force_sum) {
          case AccumulationMethod::SPLIT:
            switch (eval_energy) {
            case EvaluateEnergy::YES:
              ktgfsGBNeckForceEnergyNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                                             clash_minimum_distance, clash_ratio,
                                                             *scw, *iswk, *gmem_r);
              break;
            case EvaluateEnergy::NO:
              ktgfsGBNeckForceNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                                       clash_minimum_distance, clash_ratio, *iswk,
                                                       *gmem_r);
              break;
            }
            ktgfsCalculateGBNeckDerivatives<<<gbd_bt.x, gbd_bt.y>>>(poly_nbk, *ctrl, *poly_psw,
                                                                    *iswk, *gmem_r);
            break;
          case AccumulationMethod::WHOLE:
            switch (eval_energy) {
            case EvaluateEnergy::YES:
              ktgfGBNeckForceEnergyNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                                            clash_minimum_distance, clash_ratio,
                                                            *scw, *iswk, *gmem_r);
              break;
            case EvaluateEnergy::NO:
              ktgfGBNeckForceNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                                      clash_minimum_distance, clash_ratio, *iswk,
                                                      *gmem_r);
              break;
            }
            ktgfCalculateGBNeckDerivatives<<<gbd_bt.x, gbd_bt.y>>>(poly_nbk, *ctrl, *poly_psw,
                                                                   *iswk, *gmem_r);
            break;
          case AccumulationMethod::AUTOMATIC:
            break;
          }
          break;
        case EvaluateForce::NO:
          ktgfGBNeckEnergyNonClash<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw,
                                                   clash_minimum_distance, clash_ratio, *scw,
                                                   *iswk, *gmem_r);
          break;
        }
        break;
      }
      break;
    case NbwuKind::SUPERTILES:
    case NbwuKind::HONEYCOMB:
      break;
    }
  }
  else {
    switch (kind) {
    case NbwuKind::TILE_GROUPS:
      switch (poly_nbk.igb) {
      case ImplicitSolventModel::NONE:
        switch (eval_force) {
        case EvaluateForce::YES:
          switch (actual_force_sum) {
          case AccumulationMethod::SPLIT:
            switch (eval_energy) {
            case EvaluateEnergy::YES:
              ktgfsVacuumForceEnergy<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *scw,
                                                     *tstw, *gmem_r);
              break;
            case EvaluateEnergy::NO:
              ktgfsVacuumForce<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *tstw,
                                               *gmem_r);
              break;
            }
            break;
          case AccumulationMethod::WHOLE:
            switch (eval_energy) {
            case EvaluateEnergy::YES:
              ktgfVacuumForceEnergy<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *scw,
                                                    *tstw, *gmem_r);
              break;
            case EvaluateEnergy::NO:
              ktgfVacuumForce<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *tstw,
                                              *gmem_r);
              break;
            }
            break;
          case AccumulationMethod::AUTOMATIC:

            // This case was converted into SPLIT or WHOLE by evaluating actual_force_sum
            break;
          }
          break;
        case EvaluateForce::NO:
          ktgfVacuumEnergy<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *scw, *tstw,
                                           *gmem_r);
          break;
        }
        break;
      case ImplicitSolventModel::HCT_GB:
      case ImplicitSolventModel::OBC_GB:
      case ImplicitSolventModel::OBC_GB_II:
        switch (actual_force_sum) {
        case AccumulationMethod::SPLIT:
          ktgfsCalculateGBRadii<<<gbr_bt.x, gbr_bt.y>>>(poly_nbk, *ctrl, *poly_psw, *tstw, *iswk,
                                                        *gmem_r);
          break;
        case AccumulationMethod::WHOLE:
          ktgfCalculateGBRadii<<<gbr_bt.x, gbr_bt.y>>>(poly_nbk, *ctrl, *poly_psw, *tstw, *iswk,
                                                       *gmem_r);
          break;
        case AccumulationMethod::AUTOMATIC:
          break;
        }
        switch (eval_force) {
        case EvaluateForce::YES:
          switch (actual_force_sum) {
          case AccumulationMethod::SPLIT:
            switch (eval_energy) {
            case EvaluateEnergy::YES:
              ktgfsGBForceEnergy<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *scw, *iswk,
                                                 *gmem_r);
              break;
            case EvaluateEnergy::NO:
              ktgfsGBForce<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *iswk, *gmem_r);
              break;
            }
            ktgfsCalculateGBDerivatives<<<gbd_bt.x, gbd_bt.y>>>(poly_nbk, *ctrl, *poly_psw, *iswk,
                                                                *gmem_r);
            break;
          case AccumulationMethod::WHOLE:
            switch (eval_energy) {
            case EvaluateEnergy::YES:
              ktgfGBForceEnergy<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *scw, *iswk,
                                                *gmem_r);
              break;
            case EvaluateEnergy::NO:
              ktgfGBForce<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *iswk, *gmem_r);
              break;
            }
            ktgfCalculateGBDerivatives<<<gbd_bt.x, gbd_bt.y>>>(poly_nbk, *ctrl, *poly_psw, *iswk,
                                                               *gmem_r);
            break;
          case AccumulationMethod::AUTOMATIC:
            break;
          }
          break;
        case EvaluateForce::NO:
          ktgfGBEnergy<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *scw, *iswk, *gmem_r);
          break;
        }
        break;
      case ImplicitSolventModel::NECK_GB:
      case ImplicitSolventModel::NECK_GB_II:
        switch (actual_force_sum) {
        case AccumulationMethod::SPLIT:
          ktgfsCalculateGBNeckRadii<<<gbr_bt.x, gbr_bt.y>>>(poly_nbk, *ctrl, *poly_psw, *tstw,
                                                            *iswk, *gmem_r);
          break;
        case AccumulationMethod::WHOLE:
          ktgfCalculateGBNeckRadii<<<gbr_bt.x, gbr_bt.y>>>(poly_nbk, *ctrl, *poly_psw, *tstw,
                                                           *iswk, *gmem_r);
          break;
        case AccumulationMethod::AUTOMATIC:
          break;
        }
        switch (eval_force) {
        case EvaluateForce::YES:
          switch (actual_force_sum) {
          case AccumulationMethod::SPLIT:
            switch (eval_energy) {
            case EvaluateEnergy::YES:
              ktgfsGBNeckForceEnergy<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *scw,
                                                     *iswk, *gmem_r);
              break;
            case EvaluateEnergy::NO:
              ktgfsGBNeckForce<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *iswk,
                                               *gmem_r);
              break;
            }
            ktgfsCalculateGBNeckDerivatives<<<gbd_bt.x, gbd_bt.y>>>(poly_nbk, *ctrl, *poly_psw,
                                                                    *iswk, *gmem_r);
            break;
          case AccumulationMethod::WHOLE:
            switch (eval_energy) {
            case EvaluateEnergy::YES:
              ktgfGBNeckForceEnergy<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *scw,
                                                    *iswk, *gmem_r);
              break;
            case EvaluateEnergy::NO:
              ktgfGBNeckForce<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *iswk,
                                              *gmem_r);
              break;
            }
            ktgfCalculateGBNeckDerivatives<<<gbd_bt.x, gbd_bt.y>>>(poly_nbk, *ctrl, *poly_psw,
                                                                   *iswk, *gmem_r);
            break;
          case AccumulationMethod::AUTOMATIC:
            break;
          }
          break;
        case EvaluateForce::NO:
          ktgfGBNeckEnergy<<<bt.x, bt.y>>>(poly_nbk, poly_ser, *ctrl, *poly_psw, *scw, *iswk,
                                           *gmem_r);
          break;
        }
        break;
      }
      break;
    case NbwuKind::SUPERTILES:
    case NbwuKind::HONEYCOMB:
      break;
    }
  }
}

//-------------------------------------------------------------------------------------------------
extern void launchNonbonded(const PrecisionModel prec, const AtomGraphSynthesis &poly_ag,
                            const StaticExclusionMaskSynthesis &poly_se,
                            MolecularMechanicsControls *mmctrl, PhaseSpaceSynthesis *poly_ps,
                            Thermostat *heat_bath, ScoreCard *sc, CacheResource *tb_space,
                            ImplicitSolventWorkspace *ism_space, const EvaluateForce eval_force,
                            const EvaluateEnergy eval_energy, const AccumulationMethod force_sum,
                            const CoreKlManager &launcher, const double clash_minimum_distance,
                            const double clash_ratio) {
  const HybridTargetLevel tier = HybridTargetLevel::DEVICE;
  PsSynthesisWriter poly_psw = poly_ps->data(tier);
  ScoreCardWriter scw = sc->data(tier);
  const SeMaskSynthesisReader poly_ser = poly_se.data();
  const NbwuKind nb_work_type = poly_ag.getNonbondedWorkType();
  const ImplicitSolventModel ism_type = poly_ag.getImplicitSolventModel();
  const int2 bt = launcher.getNonbondedKernelDims(prec, nb_work_type, eval_force, eval_energy,
                                                  force_sum, ism_type, ClashResponse::NONE);
  const int2 gbr_bt = launcher.getBornRadiiKernelDims(prec, nb_work_type, force_sum, ism_type);
  const int2 gbd_bt = launcher.getBornDerivativeKernelDims(prec, nb_work_type, force_sum,
                                                           ism_type);
  switch (prec) {
  case PrecisionModel::DOUBLE:
    {
      const SyNonbondedKit<double,
                           double2> poly_nbk = poly_ag.getDoublePrecisionNonbondedKit(tier);
      MMControlKit<double> ctrl = mmctrl->dpData(tier);
      CacheResourceKit<double> gmem_r = tb_space->dpData(tier);
      ThermostatWriter<double> tstw = heat_bath->dpData(tier);
      ISWorkspaceKit<double> iswk = ism_space->dpData(poly_ps->getCyclePosition(), tier);
      launchNonbonded(nb_work_type, poly_nbk, poly_ser, &ctrl, &poly_psw, &tstw, &scw, &gmem_r,
                      &iswk, eval_force, eval_energy, bt, gbr_bt, gbd_bt, clash_minimum_distance,
                      clash_ratio);
    }
    break;
  case PrecisionModel::SINGLE:
    {
      const SyNonbondedKit<float,
                           float2> poly_nbk = poly_ag.getSinglePrecisionNonbondedKit(tier);
      MMControlKit<float> ctrl = mmctrl->spData(tier);
      CacheResourceKit<float> gmem_r = tb_space->spData(tier);
      ThermostatWriter<float> tstw = heat_bath->spData(tier);
      ISWorkspaceKit<float> iswk = ism_space->spData(poly_ps->getCyclePosition(), tier);
      launchNonbonded(nb_work_type, poly_nbk, poly_ser, &ctrl, &poly_psw, &tstw, &scw, &gmem_r,
                      &iswk, eval_force, eval_energy, force_sum, bt, gbr_bt, gbd_bt,
                      clash_minimum_distance, clash_ratio);
    }
    break;
  }
}

//-------------------------------------------------------------------------------------------------
extern void launchNonbonded(const PrecisionModel prec, const AtomGraphSynthesis &poly_ag,
                            const StaticExclusionMaskSynthesis &poly_se,
                            MolecularMechanicsControls *mmctrl, PhaseSpaceSynthesis *poly_ps,
                            Thermostat *heat_bath, ScoreCard *sc, CacheResource *tb_space,
                            ImplicitSolventWorkspace *ism_space, const EvaluateForce eval_force,
                            const EvaluateEnergy eval_energy, const CoreKlManager &launcher,
                            const double clash_minimum_distance, const double clash_ratio) {
  if (prec == PrecisionModel::DOUBLE || poly_ps->getForceAccumulationBits() <= 24) {
    launchNonbonded(prec, poly_ag, poly_se, mmctrl, poly_ps, heat_bath, sc, tb_space, ism_space,
                    eval_force, eval_energy, AccumulationMethod::SPLIT, launcher,
                    clash_minimum_distance, clash_ratio);
  }
  else {
    launchNonbonded(prec, poly_ag, poly_se, mmctrl, poly_ps, heat_bath, sc, tb_space, ism_space,
                    eval_force, eval_energy, AccumulationMethod::WHOLE, launcher,
                    clash_minimum_distance, clash_ratio);
  }
}

} // namespace energy
} // namespace stormm
