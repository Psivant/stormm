#include "hip/hip_runtime.h"
// -*-c++-*-
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <limits.h>
#include <string>
#include <vector>
#include "copyright.h"
#include "../../src/Accelerator/gpu_details.h"
#include "../../src/Accelerator/hpc_config.h"
#include "../../src/Accelerator/hybrid.h"
#include "../../src/Constants/behavior.h"
#include "../../src/DataTypes/common_types.h"
#include "../../src/Namelists/command_line_parser.h"
#include "../../src/Parsing/parse.h"
#include "../../src/Random/random.h"
#include "../../src/Reporting/summary_file.h"
#include "../../src/UnitTesting/test_environment.h"
#include "../../src/UnitTesting/test_system_manager.h"
#include "../../src/UnitTesting/stopwatch.h"
#include "../../src/UnitTesting/unit_test.h"

using namespace stormm::card;
using namespace stormm::constants;
using namespace stormm::data_types;
using namespace stormm::namelist;
using namespace stormm::parse;
using namespace stormm::random;
using namespace stormm::review;
using namespace stormm::testing;

//-------------------------------------------------------------------------------------------------
// Multiply a block of data by a constant.
//
// Arguments:
//   data:    Pointer to the array to scale
//   n:       Trusted length of data
//   mult:    Factor by which to multiply all elements of the array
//-------------------------------------------------------------------------------------------------
template <typename T>
__global__ void __launch_bounds__(large_block_size, 1)
kMult(T* data, const size_t n, const T mult) {
  size_t pos = threadIdx.x + (blockIdx.x * blockDim.x);
  while (pos < n) {
    data[pos] *= mult;
    pos += (blockDim.x * gridDim.x);
  }
}

//-------------------------------------------------------------------------------------------------
// Lay out an FFT of a particular size and perform it for a given number of repeats using in-place
// transformations.  Time the result.
//
// Arguments:
//   nx:      Dimension of the FFT along the unit cell A axis
//   ny:      Dimension of the FFT along the unit cell B axis
//   nz:      Dimension of the FFT along the unit cell C axis
//   nbatch:  The number of FFTs to combine into a batch
//   prec:    Precision in which to perform calculations
//   use_ip:  Set to TRUE to apply in-place transforms, FALSE for out-of-place transforms
//   chkdir:  Set to TRUE to have forward and backward FFTs independently timed, if the problem
//            size and number of iterations is reasonable
//   xrs:     Random number generator to use in preparing data
//   iter:    The number of times to repeat the FFT / inverse FFT cycle
//   timer:   Tracks the wall time
//   gpu:     Details of the available GPU
//-------------------------------------------------------------------------------------------------
template <typename T>
void runFFT(const int nx, const int ny, const int nz, const int nbatch, const bool use_ip,
            const bool chkdir, Xoshiro256ppGenerator *xrs, const int iter, StopWatch *timer,
            const GpuDetails &gpu) {
  const PrecisionModel prec = (std::type_index(typeid(T)).hash_code() == double_type_index) ?
                              PrecisionModel::DOUBLE : PrecisionModel::SINGLE;
  const std::string place_str = (use_ip) ? "IP" : "OOP";
  const int t_id = timer->addCategory(std::string("FFT(") + intToString(nx, 3) + ", " +
                                      intToString(ny, 3) + ", " + intToString(nz, 3) + "), " +
                                      getEnumerationName(prec) + ", " + place_str);
  const int m_id = timer->addCategory(std::string("Scl(") + intToString(nx, 3) + ", " +
                                      intToString(ny, 3) + ", " + intToString(nz, 3) + "), " +
                                      getEnumerationName(prec) + ", " + place_str);
  const HybridTargetLevel devc_tier = HybridTargetLevel::DEVICE;
  const int nxyz     = nx * ny * nz;
  const int nz_pad   = 2 * ((nz / 2) + 1);
  const int nz_eff   = (use_ip) ? nz_pad : nz;
  const int nxyz_eff = nx * ny * nz_eff;
  std::vector<double> trial_load = gaussianRand(xrs, nxyz * nbatch, 1.0);
  hipfftHandle frwd_plan, bkwd_plan;
  bool frwd_problem = false;
  bool bkwd_problem = false;

  // The following variables are computed for batch FFT setup.
  int dims[] = { nx, ny, nz };
  int real_embed[3];
  real_embed[0] = nx;
  real_embed[1] = ny;
  real_embed[2] = nz_eff;
  int cmpx_embed[3];
  cmpx_embed[0] = nx;
  cmpx_embed[1] = ny;
  cmpx_embed[2] = (use_ip) ? nz_pad / 2 : nz;
  const int real_length = real_embed[0] * real_embed[1] * real_embed[2];
  const int cmpx_length = cmpx_embed[0] * cmpx_embed[1] * cmpx_embed[2];
  const std::string batch_msg = (nbatch == 1) ? ")" : "), batch " + std::to_string(nbatch);
  const double nxyz_inv_scale = 1.0 / static_cast<double>(nxyz);

  // Create the FFT plans
  hipfftType frwd_kind, bkwd_kind;
  switch (prec) {
  case PrecisionModel::DOUBLE:
    frwd_kind = HIPFFT_D2Z;
    bkwd_kind = HIPFFT_Z2D;
    break;
  case PrecisionModel::SINGLE:
    frwd_kind = HIPFFT_R2C;
    bkwd_kind = HIPFFT_C2R;
    break;
  }
  if (nbatch == 1) {
    if (hipfftPlan3d(&frwd_plan, nx, ny, nz, frwd_kind) != HIPFFT_SUCCESS) {
      frwd_problem = true;
    }
    if (hipfftPlan3d(&bkwd_plan, nx, ny, nz, bkwd_kind) != HIPFFT_SUCCESS) {
      bkwd_problem = true;
    }
  }
  else {
    if (hipfftPlanMany(&frwd_plan, 3, dims, real_embed, 1, real_length, cmpx_embed, 1,
                      cmpx_length, frwd_kind, nbatch) != HIPFFT_SUCCESS) {
      frwd_problem = true;
    }
    if (hipfftPlanMany(&bkwd_plan, 3, dims, cmpx_embed, 1, cmpx_length, real_embed, 1,
                      real_length, bkwd_kind, nbatch) != HIPFFT_SUCCESS) {
      bkwd_problem = true;
    }
  }
  if (frwd_problem || bkwd_problem) {
    const std::string pprob = (frwd_problem) ? "forward" : "inverse";
    rtErr("Failed to create " + pprob + " plan for " + std::to_string(nx) + " x " +
          std::to_string(ny) + " x " + std::to_string(nz) + " points (" +
          getEnumerationName(prec) + batch_msg + ".", "runFFT");
  }
  const int nblocks  = gpu.getSMPCount();
  const int nthreads = large_block_size;

  // Allocate and fill a data array, upload, and perform the FFT cycle 100x
  Hybrid<T> trial(nbatch * nxyz_eff, "trial_fft");
  Hybrid<T> trial_t(HybridKind::ARRAY, "trial_fft_t");
  T* trial_ptr = trial.data();
  for (int n = 0; n < nbatch; n++) {
    for (int i = 0; i < nx; i++) {
      for (int j = 0; j < ny; j++) {
        const int nij_t = (n * nxyz_eff) + (((i * ny) + j) * nz_eff);
        const int nij   = (n * nxyz)     + (((i * ny) + j) * nz);
        for (int k = 0; k < nz; k++) {
          trial_ptr[nij_t + k] = trial_load[nij + k];
        }
      }
    }
  }
  trial.upload();
  timer->assignTime(0);

  // Time the scaling kernel--this must be applied to ensure that the data does not become full of
  // NaN or Inf values after many iterations.
  for (int i = 0; i < iter; i++) {
    kMult<T><<<nblocks, nthreads>>>(trial.data(devc_tier), nbatch * real_length, 1.0);
  }
  if (hipDeviceSynchronize() != hipSuccess) {
    rtErr("CUDA device synchronize failed.", "runIpFFT");
  }
  timer->assignTime(m_id);

  // Set pointers to the data.  Pairs of pointers for both single- and double-precision data will
  // be created, but only one pair will be valid and used.
  hipfftDoubleReal* real_data = reinterpret_cast<hipfftDoubleReal*>(trial.data(devc_tier));
  hipfftReal* sp_real_data    = reinterpret_cast<hipfftReal*>(trial.data(devc_tier));
  hipfftDoubleComplex* cmpx_data;
  hipfftComplex* sp_cmpx_data;
  if (use_ip) {
    cmpx_data    = reinterpret_cast<hipfftDoubleComplex*>(trial.data(devc_tier));        
    sp_cmpx_data = reinterpret_cast<hipfftComplex*>(trial.data(devc_tier));        
  }
  else {
    trial_t.resize(2 * nbatch * nx * ny * nz);
    cmpx_data = reinterpret_cast<hipfftDoubleComplex*>(trial_t.data(devc_tier));
    sp_cmpx_data = reinterpret_cast<hipfftComplex*>(trial_t.data(devc_tier));        
  }

  // Run the FFT for the required number of iterations.  Rescale the result if needed.
  timer->assignTime(0);
  for (int i = 0; i < iter; i++) {
    switch (prec) {
    case PrecisionModel::DOUBLE:
      if (hipfftExecD2Z(frwd_plan, real_data, cmpx_data) != HIPFFT_SUCCESS) {
        frwd_problem = true;
        break;
      }
      if (hipfftExecZ2D(bkwd_plan, cmpx_data, real_data) != HIPFFT_SUCCESS) {
        bkwd_problem = true;
        break;
      }
      break;
    case PrecisionModel::SINGLE:
      if (hipfftExecR2C(frwd_plan, sp_real_data, sp_cmpx_data) != HIPFFT_SUCCESS) {
        frwd_problem = true;
        break;
      }
      if (hipfftExecC2R(bkwd_plan, sp_cmpx_data, sp_real_data) != HIPFFT_SUCCESS) {
        bkwd_problem = true;
        break;
      }
      break;
    }
    kMult<T><<<nblocks, nthreads>>>(trial.data(devc_tier), nbatch * real_length, nxyz_inv_scale);
  }
  
  // Synchronize the CPU to get timings on the FFT
  if (hipDeviceSynchronize() != hipSuccess) {
    rtErr("CUDA device synchronize failed.", "runIpFFT");
  }
  timer->assignTime(t_id);
  if (frwd_problem || bkwd_problem) {
    const std::string pprob = (frwd_problem) ? "forward" : "inverse";
    rtErr("Failed to execute " + pprob + " FFT for " + std::to_string(nx) + " x " +
          std::to_string(ny) + " x " + std::to_string(nz) + " points (" +
          getEnumerationName(prec) + batch_msg + ".", "runIpFFT");
  }

  // Check the results, if it is reasonable to do so.  Many iterations of the FFT may create some
  // degree of wander in the output, but working with just a few iterations should make it possible
  // to recover the original data to within some fairly tight bounds.
  trial.download();
  std::vector<double> trial_outcome(nbatch * nxyz);
  for (int n = 0; n < nbatch; n++) {
    for (int i = 0; i < nx; i++) {
      for (int j = 0; j < ny; j++) {
        const int nij_t = (n * nxyz_eff) + (((i * ny) + j) * nz_eff);
        const int nij   = (n * nxyz)     + (((i * ny) + j) * nz);
        for (int k = 0; k < nz; k++) {
          trial_outcome[nij + k] = trial_ptr[nij_t + k];
        }
      }
    }
  }
  if (iter < 5) {
    const double tol = static_cast<double>(iter) * 1.0e-5;
    check(trial_outcome, RelationalOperator::EQUAL, Approx(trial_load).margin(tol), "The result "
          "of forward and inverse FFTs on " + std::to_string(nx) + " x " + std::to_string(ny) +
          " x " + std::to_string(nz) + " points did not recover the original data.  Precision "
          "model: " + getEnumerationName(prec) + ".  Batch count: " + std::to_string(nbatch) +
          ".");
  }

  // Check the forward and backward FFT timings independently
  if (nx == ny && nx == nz && nbatch == 1 && iter < (INT_MAX / (nx * ny * nz)) - 1 &&
      iter < 100 && chkdir) {
    Hybrid<T> x_trial(iter * nx * ny * nz_eff, "replica_trial");
    Hybrid<T> x_trial_t(HybridKind::ARRAY, "replica_trial_t");
    hipfftDoubleReal* x_real_data = reinterpret_cast<hipfftDoubleReal*>(x_trial.data(devc_tier));
    hipfftReal* sp_x_real_data    = reinterpret_cast<hipfftReal*>(x_trial.data(devc_tier));
    hipfftDoubleComplex* x_cmpx_data;
    hipfftComplex* sp_x_cmpx_data;
    if (use_ip) {
      x_cmpx_data    = reinterpret_cast<hipfftDoubleComplex*>(x_trial.data(devc_tier));        
      sp_x_cmpx_data = reinterpret_cast<hipfftComplex*>(x_trial.data(devc_tier));        
    }
    else {
      x_trial_t.resize(2 * iter * nx * ny * nz);
      x_cmpx_data = reinterpret_cast<hipfftDoubleComplex*>(x_trial_t.data(devc_tier));
      sp_x_cmpx_data = reinterpret_cast<hipfftComplex*>(x_trial_t.data(devc_tier));        
    }
    T* x_trial_ptr = x_trial.data();
    for (int n = 0; n < iter; n++) {
      for (int i = 0; i < nx; i++) {
        for (int j = 0; j < ny; j++) {
          const int nij_t = (n * nxyz_eff) + (((i * ny) + j) * nz_eff);
          for (int k = 0; k < nz; k++) {
            x_trial_ptr[nij_t + k] = xrs->gaussianRandomNumber();
          }
        }
      }
    }
    x_trial.upload();
    const int frwd_id = timer->addCategory(std::string("Fwd(") + intToString(nx, 3) + ", " +
                                           intToString(ny, 3) + ", " + intToString(nz, 3) + "), " +
                                           getEnumerationName(prec) + ", " + place_str);
    const int bkwd_id = timer->addCategory(std::string("Inv(") + intToString(nx, 3) + ", " +
                                           intToString(ny, 3) + ", " + intToString(nz, 3) + "), " +
                                           getEnumerationName(prec) + ", " + place_str);
    timer->assignTime(0);
    for (int i = 0; i < iter; i++) {
      const size_t real_idx = i * nxyz_eff;
      const size_t cmpx_idx = (use_ip) ? i * nxyz_eff / 2 : i * nxyz_eff;
      switch (prec) {
      case PrecisionModel::DOUBLE:
        if (hipfftExecD2Z(frwd_plan,
                         &x_real_data[real_idx], &x_cmpx_data[cmpx_idx]) != HIPFFT_SUCCESS) {
          frwd_problem = true;
          break;
        }
        break;
      case PrecisionModel::SINGLE:
        if (hipfftExecR2C(frwd_plan,
                         &sp_x_real_data[real_idx], &sp_x_cmpx_data[cmpx_idx]) != HIPFFT_SUCCESS) {
          frwd_problem = true;
          break;
        }
        break;
      }
    }
    if (hipDeviceSynchronize() != hipSuccess) {
      rtErr("CUDA device synchronize failed.", "runIpFFT");
    }
    timer->assignTime(frwd_id);
    for (int i = 0; i < iter; i++) {
      const size_t real_idx = i * nxyz_eff;
      const size_t cmpx_idx = (use_ip) ? i * nxyz_eff / 2 : i * nxyz_eff;
      switch (prec) {
      case PrecisionModel::DOUBLE:
        if (hipfftExecZ2D(bkwd_plan,
                         &x_cmpx_data[cmpx_idx], &x_real_data[real_idx]) != HIPFFT_SUCCESS) {
          bkwd_problem = true;
          break;
        }
        break;
      case PrecisionModel::SINGLE:
        if (hipfftExecC2R(bkwd_plan,
                         &sp_x_cmpx_data[cmpx_idx], &sp_x_real_data[real_idx]) != HIPFFT_SUCCESS) {
          bkwd_problem = true;
          break;
        }
        break;
      }
    }
    if (hipDeviceSynchronize() != hipSuccess) {
      rtErr("CUDA device synchronize failed.", "runIpFFT");
    }
    timer->assignTime(bkwd_id);
  }
  
  // Destroy the plans to prevent having too many active handles
  if (hipfftDestroy(frwd_plan) != HIPFFT_SUCCESS) {
    frwd_problem = true;
  }
  if (hipfftDestroy(bkwd_plan) != HIPFFT_SUCCESS) {
    bkwd_problem = true;
  }
  if (frwd_problem || bkwd_problem) {
    const std::string pprob = (frwd_problem) ? "forward" : "inverse";
    rtErr("Failed to destroy the " + pprob + " FFT plan for " + std::to_string(nx) + " x " +
          std::to_string(ny) + " x " + std::to_string(nz) + " points (" +
          getEnumerationName(prec) + batch_msg + ".", "runIpFFT");
  }
}

//-------------------------------------------------------------------------------------------------
// Perform a simple, out-of-place, round-trip FFT for an 8 x 8 x 8 grid and check the result.
//
// Arguments:
//   xrs:     Random number generator to produce initial data
//-------------------------------------------------------------------------------------------------
void simpleOopFFT(Xoshiro256ppGenerator *xrs) {
  Hybrid<double> trial(512, "trial_fft"), trial_t(1024, "trial_fft_t");
  const std::vector<double> trial_fill = gaussianRand(xrs, trial.size(), 1.0);
  trial.putHost(trial_fill);
  trial.upload();
  hipfftHandle trial_plan, trial_plan_inv;
  if (hipfftPlan3d(&trial_plan, 8, 8, 8, HIPFFT_D2Z) != HIPFFT_SUCCESS) {
    rtErr("Unable to create forward plan for the trial.", "main");
  }
  if (hipfftPlan3d(&trial_plan_inv, 8, 8, 8, HIPFFT_Z2D) != HIPFFT_SUCCESS) {
    rtErr("Unable to create inverse plan for the trial.", "main");
  }
  hipfftExecD2Z(trial_plan,
               reinterpret_cast<hipfftDoubleReal*>(trial.data(HybridTargetLevel::DEVICE)),
               reinterpret_cast<hipfftDoubleComplex*>(trial_t.data(HybridTargetLevel::DEVICE)));
  hipfftExecZ2D(trial_plan_inv,
               reinterpret_cast<hipfftDoubleComplex*>(trial_t.data(HybridTargetLevel::DEVICE)),
               reinterpret_cast<hipfftDoubleReal*>(trial.data(HybridTargetLevel::DEVICE)));
  hipfftDestroy(trial_plan);
  hipfftDestroy(trial_plan_inv);
  std::vector<double> trial_result = trial.readDevice();
  for (int i = 0; i < 512; i++) {
    trial_result[i] /= 512.0;
  }
  check(trial_result, RelationalOperator::EQUAL, trial_fill, "The result of forward and inverse "
        "out-of-place FFTs does not match the original input.");
}

//-------------------------------------------------------------------------------------------------
// Perform a simple, in-place, round-trip FFT for an 8 x 8 x 8 grid and check the result.
//
// Arguments:
//   xrs:     Random number generator to produce initial data
//-------------------------------------------------------------------------------------------------
void simpleIpFFT(Xoshiro256ppGenerator *xrs) {
  const int nx = 8;
  const int ny = 8;
  const int nz = 8;
  const int nz_pad = 2 * ((nz / 2) + 1);
  Hybrid<double> trial(nx * ny * nz_pad, "trial_fft");
  const std::vector<double> trial_fill = gaussianRand(xrs, nx * ny * nz, 1.0);
  double* trial_ptr = trial.data();
  for (int i = 0; i < nx; i++) {
    for (int j = 0; j < ny; j++) {
      for (int k = 0; k < nz; k++) {
        trial_ptr[(((i * ny) + j) * nz_pad) + k] = trial_fill[(((i * ny) + j) * nz) + k];
      }
    }
  }
  trial.upload();
  hipfftHandle trial_plan, trial_plan_inv;
  if (hipfftPlan3d(&trial_plan, 8, 8, 8, HIPFFT_D2Z) != HIPFFT_SUCCESS) {
    rtErr("Unable to create forward plan for the trial.", "main");
  }
  if (hipfftPlan3d(&trial_plan_inv, 8, 8, 8, HIPFFT_Z2D) != HIPFFT_SUCCESS) {
    rtErr("Unable to create inverse plan for the trial.", "main");
  }
  hipfftExecD2Z(trial_plan,
               reinterpret_cast<hipfftDoubleReal*>(trial.data(HybridTargetLevel::DEVICE)),
               reinterpret_cast<hipfftDoubleComplex*>(trial.data(HybridTargetLevel::DEVICE)));
  hipfftExecZ2D(trial_plan_inv,
               reinterpret_cast<hipfftDoubleComplex*>(trial.data(HybridTargetLevel::DEVICE)),
               reinterpret_cast<hipfftDoubleReal*>(trial.data(HybridTargetLevel::DEVICE)));
  hipfftDestroy(trial_plan);
  hipfftDestroy(trial_plan_inv);
  const std::vector<double> trial_result_raw = trial.readDevice();
  std::vector<double> trial_result(nx * ny * nz);
  for (int i = 0; i < nx; i++) {
    for (int j = 0; j < ny; j++) {
      for (int k = 0; k < nz; k++) {
        trial_result[(((i * ny) + j) * nz) + k] = trial_result_raw[(((i * ny) + j) * nz_pad) + k];
      }
    }
  }
  for (int i = 0; i < 512; i++) {
    trial_result[i] /= 512.0;
  }
  check(trial_result, RelationalOperator::EQUAL, trial_fill, "The result of forward and inverse "
        "in-place FFTs does not match the original input.");
}

//-------------------------------------------------------------------------------------------------
// main
//-------------------------------------------------------------------------------------------------
int main(const int argc, const char* argv[]) {

  // Baseline variables

  // Parse command-line information
  CommandLineParser clip("fft_throughput", "A benchmarking program for measuring the processing "
                         "time of various 3D Fast Fourier Transforms evaluated by the NVIDIA "
                         "cuFFT library.  The program is designed to test a range of problem "
                         "sizes relevant to PME molecular dynamics simulations.", { "-timings" });
  clip.activateHelpOnNoArgs();
  clip.activateExitOnHelp();
  NamelistEmulator *t_nml = clip.getNamelistPointer(); 
  t_nml->addKeyword("-ip", NamelistType::BOOLEAN);
  t_nml->addHelp("-ip", "Use in-place 3D FFTs.");
  t_nml->addKeyword("-double", NamelistType::BOOLEAN);
  t_nml->addHelp("-double", "Request double-precision 3D FFT calculations.  This is many times "
                 "more expensive than single-precision calculations on most NVIDIA cards, about "
                 "twice as expensive on cards of the X100 line.");
  t_nml->addKeyword("-single", NamelistType::BOOLEAN);
  t_nml->addHelp("-single", "Request single-precision 3D FFT calculations.  If neither -double "
                 "nor -single is specified, single-precision calculations will be scheduled.");
  t_nml->addKeyword("-directional", NamelistType::BOOLEAN);
  t_nml->addHelp("-directional", "Check both forward and iverse 3D FFTs of the selected sizes.");
  t_nml->addKeyword("-batch", NamelistType::INTEGER, std::to_string(1));
  t_nml->addHelp("-batch", "The number of FFTs of each selected size to batch in a single "
                 "calculation.");
  t_nml->addKeyword("-iter", NamelistType::INTEGER, std::to_string(100));
  t_nml->addHelp("-iter", "The number of iterations with which to perform each FFT cycle.  The "
                 "contents of each mesh (populated with random numbers) will be re-normalized "
                 "after each calculation to prevent the numbers from growing in Inf or NaN.  The "
                 "time to re-normalize each grid will be pre-calculated and must then be "
                 "subtracted automatically from the FFT result.");
  t_nml->addKeyword("-max_grid", NamelistType::INTEGER, std::to_string(256));
  t_nml->addHelp("-max_grid", "The maximum size of the FFT problem mesh grid along any one side.  "
                 "Up to this value, a range of sizes which factorize into 2, 3, 5, 7, and 11 will "
                 "be used to create mesh grids.");
  t_nml->addKeyword("-min_grid", NamelistType::INTEGER, std::to_string(8));
  t_nml->addHelp("-min_grid", "The minimum size of the FFT problem mesh grid along any one side.");
  t_nml->addKeyword("-max_pts", NamelistType::INTEGER, std::to_string(1024 * 1024 * 1024));
  t_nml->addHelp("-max_pts", "The maximum number of grid points that will be allowed in any one "
                 "problem, including batches of multiple grids.  For example, if the maximum "
                 "number of points is 1 million, a 3D FFTs of size 100 x 100 x 100, 100 x 50 x "
                 "200, or 125 x 75 x 100 would be permissible.  A batch of four FFTS of size 100 "
                 "x 50 x 50 would likewise be attempted.");
  t_nml->addKeyword("-min_pts", NamelistType::INTEGER, std::to_string(512));
  t_nml->addHelp("-min_pts", "The minimum size of any one grid or batch of grids to profile.  See "
                 "-max_pts, above.");
  t_nml->addKeyword("-radix", NamelistType::INTEGER, std::string(""), DefaultIsObligatory::NO,
                    InputRepeats::YES);
  t_nml->setImperative("-radix", KeyRequirement::OPTIONAL);
  t_nml->addHelp("-radix", "A radix which will be required in each FFT problem.  Repeated "
                 "inputs may beused to require a particular radix more than once, whether within "
                 "one dimension of the problem or spread across all three dimensions.");
  t_nml->addKeyword("-noradix", NamelistType::INTEGER, std::string(""), DefaultIsObligatory::NO,
                    InputRepeats::YES);
  t_nml->setImperative("-noradix", KeyRequirement::OPTIONAL);
  t_nml->addHelp("-noradix", "Forbid that a particular radix be present in any FFT problem.  This "
                 "keyword may be specified repeatedly, and even mention radices that have been "
                 "required a particular number of times.  For example, specifying \"-radix 7 "
                 "-radix 7 -noradix 7\" would stipulate that each FFT problem contains exactly "
                 "two radices of 7, although they may occur along one side of the mesh or in two "
                 "out of three sides.");
  TestEnvironment oe(argc, argv, &clip, TmpdirStatus::NOT_REQUIRED, ExceptionResponse::SILENT);
  clip.parseUserInput(argc, argv);
  
  // Take in additional command-line variables
  const bool use_ip = t_nml->getBoolValue("-ip");
  const int nbatch = t_nml->getIntValue("-batch");
  const int iter = t_nml->getIntValue("-iter");
  const int max_grid = t_nml->getIntValue("-max_grid");
  const int max_pts = t_nml->getIntValue("-max_pts");
  const int min_grid = t_nml->getIntValue("-min_grid");
  const int min_pts = t_nml->getIntValue("-min_pts");
  bool test_double = t_nml->getBoolValue("-double");
  bool test_single = t_nml->getBoolValue("-single");
  bool check_directional_ffts = t_nml->getBoolValue("-directional");
  std::vector<int> radices;
  if (t_nml->getKeywordStatus("-radix") == InputStatus::USER_SPECIFIED) {
    radices = t_nml->getAllIntValues("-radix");
  }
  std::vector<int> non_radices;
  if (t_nml->getKeywordStatus("-noradix") == InputStatus::USER_SPECIFIED) {
    radices = t_nml->getAllIntValues("-noradix");
  }

  // Adjust inputs
  if (test_double == false && test_single == false) {
    test_single = true;
  }
  if (oe.getVerbosity() == TestVerbosity::FULL) {
    stormmSplash();
  }
  StopWatch timer;
#ifdef STORMM_USE_CUDA
  const HpcConfig gpu_config(ExceptionResponse::WARN);
  const std::vector<int> my_gpus = gpu_config.getGpuDevice(1);
  const GpuDetails gpu = gpu_config.getGpuInfo(my_gpus[0]);
  Hybrid<int> force_gpu_to_engage(1);
#endif

  // Initialize a random number generator and run some trials
  Xoshiro256ppGenerator xrs;
  simpleOopFFT(&xrs);
  simpleIpFFT(&xrs);
  const std::vector<int> ffts = { 24, 28, 30, 32, 33, 36, 40, 44, 45, 48, 54, 55, 56, 60, 64, 66,
                                  72, 75, 77, 80, 84, 88, 90, 96, 99, 100, 108, 110, 112, 120, 128,
                                  132, 135, 140, 144, 150, 154, 160, 165, 168, 176, 180, 192, 198,
                                  200, 210, 220, 224, 231, 240, 250, 256 };
  std::vector<PrecisionModel> all_prec;
  if (test_double) {
    all_prec.push_back(PrecisionModel::DOUBLE);
  }
  if (test_single) {
    all_prec.push_back(PrecisionModel::SINGLE);
  }
  int radix_product = 1;
  if (radices.size() > 0) {
    for (size_t i = 0; i < radices.size(); i++) {
      radix_product *= radices[i];
    }
  }
  for (size_t p = 0; p < all_prec.size(); p++) {
    for (size_t i = 0; i < ffts.size(); i++) {
      for (size_t j = i; j < ffts.size(); j++) {
        for (size_t k = j; k < ffts.size(); k++) {

          // Check that the aspect ratios are realistic for a typical MD simulation
          const int min_ij = std::min(ffts[i], ffts[j]);
          const int min_ik = std::min(ffts[i], ffts[k]);
          const int min_jk = std::min(ffts[j], ffts[k]);
          if (ffts[k] > max_grid || ffts[i] < min_grid ||
              (std::max(ffts[i], ffts[j]) + (min_ij - 1)) / min_ij > 2 ||
              (std::max(ffts[i], ffts[k]) + (min_ik - 1)) / min_ik > 2 ||
              (std::max(ffts[j], ffts[k]) + (min_jk - 1)) / min_jk > 2) {
            continue;
          }
          const int total_pts = ffts[i] * ffts[j] * ffts[k];

          // Skip if the total point count is outside the range of interest
          if (total_pts * nbatch < min_pts || total_pts * nbatch > max_pts) {
            continue;
          }

          // Skip is the requested radices are not present
          if (radix_product > 1 && (total_pts % radix_product) > 0) {
            continue;
          }

          // Skip if a forbidden radix is present, after accounting for requested radices
          const int rquot = total_pts / radix_product;
          bool has_bad_radix = false;
          for (size_t ir = 0; ir < non_radices.size(); ir++) {
            has_bad_radix = (has_bad_radix || (rquot % non_radices[ir]) == 0);
          }
          if (has_bad_radix) {
            continue;
          }
          
          // Perform the FFTs in the requested mode
          switch (all_prec[p]) {
          case PrecisionModel::DOUBLE:
            runFFT<double>(ffts[i], ffts[j], ffts[k], nbatch, use_ip, check_directional_ffts,
                           &xrs, iter, &timer, gpu);
            break;
          case PrecisionModel::SINGLE:
            runFFT<float>(ffts[i], ffts[j], ffts[k], nbatch, use_ip, check_directional_ffts,
                          &xrs, iter, &timer, gpu);
            break;
          }
        }
      }
    }
  }

  // Summary evaluation
  if (oe.getDisplayTimingsOrder()) {
    timer.assignTime(0);
    timer.printResults();
  }
  return 0;
}
