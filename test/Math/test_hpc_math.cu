#include "hip/hip_runtime.h"
// -*-c++-*-
#include "../../src/Constants/behavior.h"
#include "../../src/Constants/hpc_bounds.h"
#include "../../src/Accelerator/hpc_config.h"
#include "../../src/Accelerator/hybrid.h"
#include "../../src/FileManagement/file_listing.h"
#include "../../src/Math/bspline.h"
#include "../../src/Math/summation.h"
#include "../../src/Math/hpc_summation.cuh"
#include "../../src/Reporting/error_format.h"
#include "../../src/Random/random.h"
#include "../../src/Random/hpc_random.h"
#include "../../src/UnitTesting/unit_test.h"

using stormm::constants::ExceptionResponse;
using stormm::constants::tiny;
using stormm::constants::large_block_size;
using stormm::constants::warp_bits_mask_int;
using stormm::card::HpcConfig;
using stormm::card::Hybrid;
using stormm::card::HybridFormat;
using stormm::card::HybridTargetLevel;
using stormm::card::GpuDetails;
using stormm::data_types::llint;
using stormm::data_types::ullint;
using stormm::data_types::ullint2;
using stormm::data_types::ullint4;
using stormm::diskutil::DrivePathType;
using stormm::diskutil::getDrivePathType;
using stormm::diskutil::osSeparator;
using stormm::errors::rtWarn;
using stormm::random::max_xo_long_jumps;
using stormm::random::initXoroshiro128pArray;
using stormm::random::initXoshiro256ppArray;
using stormm::random::Ran2Generator;
using stormm::random::Xoroshiro128pGenerator;
using stormm::random::Xoshiro256ppGenerator;
using stormm::random::default_xoroshiro128p_scrub;
using stormm::random::default_xoshiro256pp_scrub;
using namespace stormm::stmath;
using namespace stormm::hpc_math;
using namespace stormm::testing;

#include "../../src/Math/bspline.cui"

//-------------------------------------------------------------------------------------------------
// Load a vector of random numbers using a CPU-based Ran2 generator.
//
// Arguments:
//   va:    The Hybrid object to load (modified and returned)
//   prng:  Ran2 generator object
//-------------------------------------------------------------------------------------------------
void loadRan2ByCPU(Hybrid<double> *va, Ran2Generator *prng) {
  double* vdata = va->data();
  const int nval = va->size();
  for (int i = 0; i < nval; i++) {
    vdata[i] = (prng->uniformRandomNumber() - 0.5) * sqrt(static_cast<double>(i + 1));
  }
}

//-------------------------------------------------------------------------------------------------
// Load a vector of random numbers using a CPU-based xoroshiro128+ generator.
//
// Arguments:
//   va:    The Hybrid object to load (modified and returned)
//   prng:  Xoroshiro128+ generator object
//-------------------------------------------------------------------------------------------------
void loadXoroshiro128pByCPU(Hybrid<double> *va, Xoroshiro128pGenerator *prng) {
  double* vdata = va->data();
  const int nval = va->size();
  for (int i = 0; i < nval; i++) {
    vdata[i] = (prng->uniformRandomNumber() - 0.5) * sqrt(static_cast<double>(i + 1));
  }
}

//-------------------------------------------------------------------------------------------------
// Load a vector of random numbers using a CPU-based xoshiro256++ generator.
//
// Arguments:
//   va:    The Hybrid object to load (modified and returned)
//   prng:  Xoroshiro128+ generator object
//-------------------------------------------------------------------------------------------------
void loadXoshiro256ppByCPU(Hybrid<double> *va, Xoshiro256ppGenerator *prng) {
  double* vdata = va->data();
  const int nval = va->size();
  for (int i = 0; i < nval; i++) {
    vdata[i] = (prng->uniformRandomNumber() - 0.5) * sqrt(static_cast<double>(i + 1));
  }
}

//-------------------------------------------------------------------------------------------------
// Evaluate random numbers from a plethora of xoroshiro128+ generator states.
//
// Arguments:
//   state_vector:  The vector of all states
//   n_generators:  Total number of states
//   samples:       Vector of states and iteration counts to watch out for
//   random_ouput:  Random numbers produced by the requested states and iterations
//   n_samples:     The number of samples (length of samples and random_output)
//-------------------------------------------------------------------------------------------------
__global__ void __launch_bounds__(large_block_size, 1)
kEvalXoroshiro128p(ullint2* state_vector, const int n_generators, const int2* samples,
                   double* random_output, const int n_samples) {
  __shared__ volatile int smax_iter;
  
  // Find the maximum iteration and cache the samples
  if (threadIdx.x == 0) {
    smax_iter = 0;
  }
  int max_iter = 0;
  for (int pos = threadIdx.x; pos < n_samples; pos += blockDim.x) {
    const int2 tmp_sample = samples[pos];
    if (tmp_sample.y > max_iter) {
      max_iter = tmp_sample.y;
    }
  }
#ifdef STORMM_USE_HIP
  const int init_shfl_stride = 32;
#else
  const int init_shfl_stride = 16;
#endif
  for (int i = init_shfl_stride; i > 0; i >>= 1) {
    const int max_comp = SHFL_DOWN(max_iter, i);
    if (max_comp > max_iter) {
      max_iter = max_comp;
    }
  }
  __syncthreads();
  const int lane_idx = (threadIdx.x & warp_bits_mask_int);
  if (lane_idx == 0 && max_iter > 0) {
    atomicMax((int*)&smax_iter, max_iter);
  }
  __syncthreads();
  const int ginc = blockDim.x * gridDim.x;
  for (int pos = threadIdx.x + (blockIdx.x * blockDim.x); pos < n_generators; pos += ginc) {
    ullint2 tmp_state = state_vector[pos];
    for (int i = 0; i <= smax_iter; i++) {

      // Get a uniform random number in the range [0, 1).  Casting the unsigned long long int to
      // a (signed) long long int works as anything great than 2^3 just becomes negative.  The
      // bit string is unchanged.
      const ullint s0 = tmp_state.x;
      ullint       s1 = tmp_state.y;
      const ullint rndbits = s0 + s1;
      const llint work = (((rndbits >> 12) & 0xfffffffffffff) | 0x3ff0000000000000);
      const double rn_out = __longlong_as_double(work) - 1.0;

      // Horribly inefficient loop and access pattern, but this is just
      // a test program.  All of this is L1-cached, too.
      for (int j = 0; j < n_samples; j++) {
        const int2 tmp_sample = samples[j];
        if (pos == tmp_sample.x && i == tmp_sample.y) {
          random_output[j] = rn_out;
        }
      }

      // Push the state forward
      s1 ^= s0;
      tmp_state.x = (((s0 << 24) | (s0 >> (64 - 24))) ^ s1 ^ (s1 << 16));
      tmp_state.y =  ((s1 << 37) | (s1 >> (64 - 37)));      
    }

    // In a simulation, the state vector would be updated after computations
    // for this thread or atom are complete.
    state_vector[pos] = tmp_state;    
  }
}

//-------------------------------------------------------------------------------------------------
// Evaluate random numbers from a plethora of xoshiro256++ generator states.
//
// Arguments:
//   state_vector:  The vector of all states
//   n_generators:  Total number of states
//   samples:       Vector of states and iteration counts to watch out for
//   random_ouput:  Random numbers produced by the requested states and iterations
//   n_samples:     The number of samples (length of samples and random_output)
//-------------------------------------------------------------------------------------------------
__global__ void __launch_bounds__(large_block_size, 1)
kEvalXoshiro256pp(ullint2* state_xy, ullint2* state_zw, const int n_generators,
                  const int2* samples, double* random_output, const int n_samples) {
  __shared__ volatile int smax_iter;
  
  // Find the maximum iteration and cache the samples
  if (threadIdx.x == 0) {
    smax_iter = 0;
  }
  int max_iter = 0;
  for (int pos = threadIdx.x; pos < n_samples; pos += blockDim.x) {
    const int2 tmp_sample = samples[pos];
    if (tmp_sample.y > max_iter) {
      max_iter = tmp_sample.y;
    }
  }
#ifdef STORMM_USE_HIP
  const int init_shfl_stride = 32;
#else
  const int init_shfl_stride = 16;
#endif
  for (int i = init_shfl_stride; i > 0; i >>= 1) {
    const int max_comp = SHFL_DOWN(max_iter, i);
    if (max_comp > max_iter) {
      max_iter = max_comp;
    }
  }
  __syncthreads();
  const int lane_idx = (threadIdx.x & warp_bits_mask_int);
  if (lane_idx == 0 && max_iter > 0) {
    atomicMax((int*)&smax_iter, max_iter);
  }
  __syncthreads();
  const int ginc = blockDim.x * gridDim.x;
  for (int pos = threadIdx.x + (blockIdx.x * blockDim.x); pos < n_generators; pos += ginc) {
    const ullint2 txy_state = state_xy[pos];
    const ullint2 tzw_state = state_zw[pos];
    ullint4 tmp_state = { txy_state.x, txy_state.y, tzw_state.x, tzw_state.y };
    for (int i = 0; i <= smax_iter; i++) {

      // Get a uniform random number in the range [0, 1).  Casting the unsigned long long int to
      // a (signed) long long int works as anything great than 2^3 just becomes negative.  The
      // bit string is unchanged.
      const ullint sxsw = tmp_state.x + tmp_state.w;
      const ullint rndbits = tmp_state.x + ((sxsw << 23) | (sxsw >> (64 - 23)));
      const llint work = (((rndbits >> 12) & 0xfffffffffffff) | 0x3ff0000000000000);
      const double rn_out = __longlong_as_double(work) - 1.0;

      // Horribly inefficient loop and access pattern, but this is just
      // a test program.  All of this is L1-cached, too.
      for (int j = 0; j < n_samples; j++) {
        const int2 tmp_sample = samples[j];
        if (pos == tmp_sample.x && i == tmp_sample.y) {
          random_output[j] = rn_out;
        }
      }

      // Push the state forward
      const ullint t = (tmp_state.y << 17);
      tmp_state.z ^= tmp_state.x;
      tmp_state.w ^= tmp_state.y;
      tmp_state.y ^= tmp_state.z;
      tmp_state.x ^= tmp_state.w;
      tmp_state.z ^= t;
      tmp_state.w = ((tmp_state.w << 45) | (tmp_state.w >> (64 - 45)));
    }

    // In a simulation, the state vector would be updated after computations
    // for this thread or atom are complete.
    state_xy[pos] = { tmp_state.x, tmp_state.y };
    state_zw[pos] = { tmp_state.z, tmp_state.w };
  }
}

//-------------------------------------------------------------------------------------------------
// Reproduce the random number generated by one of a series of xoroshiro128+ state vectors.
//
// Arguments:
//   rng_states:     List of xoroshiro128+ long-jump generator states (the dimension of this array
//                   implies the number of long-jumps taken when seeding the various states)
//   generator_idx:  Index of the generator state to query.  In a simulation, this might correspond
//                   to the atom index, or perhaps to the thread index within a particular launch
//                   grid.
//   iteration:      Produce the pseudo-random number for this point in the sequence of the
//                   particular atom or thread.
//-------------------------------------------------------------------------------------------------
double pinpointXoroshiro128p(std::vector<ullint2> &rng_states, const int generator_idx,
                             const uint iteration) {
  
  // Determine the initial generator state, possibly after having taken some short jumps.
  // The rng_states vector was created by taking up to max_xo_long_jumps long jumps of a single
  // state.  A given generator state is then created by tiling this series of initial, long jump
  // states with 1, 2, ..., n additional jumps, up to the total number of generators needed.  The
  // total number of generators is irrelevant.  This calculation requires the index for just one.
  const int n_seeds       = rng_states.size();
  const int n_short_jumps = generator_idx / n_seeds;
  const int seed_idx      = generator_idx - (n_short_jumps * n_seeds);

  // Recover the generator initial state of interest.  In a simulation, an initial generator X can
  // be jumped forward by P long jumps and N short jumps to arrive at a subsidiary generator X'.
  // Advancing X' by K iterations will produce the same output random number as advancing X by K
  // iterations, then jumping forward by P long jumps and N short jumps.
  Xoroshiro128pGenerator tgen(rng_states[seed_idx]);
  for (int i = 0; i < n_short_jumps; i++) {
    tgen.jump();
  }
  
  // Get the double-precision random number resulting from the requested iteration
  double result;
  for (uint i = 0; i <= iteration; i++) {
    result = tgen.uniformRandomNumber();
  }
  return result;
}

//-------------------------------------------------------------------------------------------------
// Reproduce the random number generated by one of a series of xoshiro256++ state vectors.
//
// Arguments:
//   rng_xy_states:  List of first halves of xoshiro256++ long-jump generator states (the dimension
//                   of this array implies the number of long-jumps taken when seeding the various
//                   states)
//   rng_zw_states:  List of second halves of xoshiro256++ long-jump generator states
//   generator_idx:  Index of the generator state to query.  In a simulation, this might correspond
//                   to the atom index, or perhaps to the thread index within a particular launch
//                   grid.
//   iteration:      Produce the pseudo-random number for this point in the sequence of the
//                   particular atom or thread.
//-------------------------------------------------------------------------------------------------
double pinpointXoshiro256pp(std::vector<ullint2> &rng_xy_states,
                            std::vector<ullint2> &rng_zw_states, const int generator_idx,
                            const uint iteration) {
  
  // Determine the initial generator state, possibly after having taken some short jumps.
  // The rng_states vector was created by taking up to max_xo_long_jumps long jumps of a single
  // state.  A given generator state is then created by tiling this series of initial, long jump
  // states with 1, 2, ..., n additional jumps, up to the total number of generators needed.  The
  // total number of generators is irrelevant.  This calculation requires the index for just one.
  const int n_seeds       = rng_xy_states.size();
  const int n_short_jumps = generator_idx / n_seeds;
  const int seed_idx      = generator_idx - (n_short_jumps * n_seeds);

  // Recover the generator initial state of interest.  In a simulation, an initial generator X can
  // be jumped forward by P long jumps and N short jumps to arrive at a subsidiary generator X'.
  // Advancing X' by K iterations will produce the same output random number as advancing X by K
  // iterations, then jumping forward by P long jumps and N short jumps.
  const ullint4 seeded_state = { rng_xy_states[seed_idx].x, rng_xy_states[seed_idx].y,
                                 rng_zw_states[seed_idx].x, rng_zw_states[seed_idx].y };
  Xoshiro256ppGenerator tgen(seeded_state);
  for (int i = 0; i < n_short_jumps; i++) {
    tgen.jump();
  }
  
  // Get the double-precision random number resulting from the requested iteration
  double result;
  for (uint i = 0; i <= iteration; i++) {
    result = tgen.uniformRandomNumber();
  }
  return result;
}

//-------------------------------------------------------------------------------------------------
// This kernel will fill in some arrays with B-spline coefficients based on values of the delta.
// Derivatives can be computed by providing a negative value of the interpolation order.
//
// Arguments:
//   dx:              The array of deltas for computing B-splines
//   n:               The trusted length of dx
//   order:           The order of B-spline coefficients to compute.  Specifying negative values
//                    will have derivatives computed.
//   coefficients:    Array of coefficients, filled and returned, ordered in stretches of values
//                    for dx(0), dx(1), ..., dx(n)
//   derivatives:     Array of derivatives, filled and returned, ordered in stretches of values
//                    for dx(0), dx(1), ..., dx(n)
//-------------------------------------------------------------------------------------------------
__global__ void __launch_bounds__(large_block_size, 1)
kCalculateBSplines(const double* dx, const int n, const int order, double* coefficients,
                   double* derivatives) {
  double bspln_knots[6], bspln_dervs[6];
  int pos = threadIdx.x + (blockIdx.x * blockDim.x);
  while (pos < n) {
    if (order == 4) {
      devcBSpline4(dx[pos], bspln_knots);
    }
    else if (order == 5) {
      devcBSpline5(dx[pos], bspln_knots);
    }
    else if (order == 6) {
      devcBSpline6(dx[pos], bspln_knots);
    }
    else if (order == -4) {
      devcBSpline4(dx[pos], bspln_knots, bspln_dervs);
    }
    else if (order == -5) {
      devcBSpline5(dx[pos], bspln_knots, bspln_dervs);
    }
    else if (order == -6) {
      devcBSpline6(dx[pos], bspln_knots, bspln_dervs);
    }

    // Transfer the results to global memory
    const int abs_order = abs(order);
    for (int i = 0; i < abs_order; i++) {
      coefficients[(pos * abs_order) + i] = bspln_knots[i];
    }
    if (order < 0) {
      for (int i = 0; i < abs_order; i++) {
        derivatives[(pos * abs_order) + i] = bspln_dervs[i];
      }
    }
    pos += (blockDim.x * gridDim.x);
  }
}

//-------------------------------------------------------------------------------------------------
// Test various B-spline device functions.
//-------------------------------------------------------------------------------------------------
void testBSplineDeviceFuncs() {
  const int npts = 2048;
  Hybrid<double> dx(npts), coefficients(6 * npts), derivatives(6 * npts);
  for (int i = 0; i < npts; i++) {
    dx.putHost((static_cast<double>(i) + 0.28) / static_cast<double>(npts), i);
  }
  dx.upload();
  const HybridTargetLevel devc_layer = HybridTargetLevel::DEVICE;
  for (int ordr = 4; ordr < 7; ordr++) {

    // Compute B-spline knots only
    kCalculateBSplines<<<1, 1024>>>(dx.data(devc_layer), npts, ordr, coefficients.data(devc_layer),
                                    derivatives.data(devc_layer));
    coefficients.download();
    
    // Check the result against the CPU function
    std::vector<double> host_knots(ordr * npts);
    double* hkn_ptr = host_knots.data();
    for (int i = 0; i < npts; i++) {
      bSpline(dx.readHost(i), ordr, &hkn_ptr[i * ordr]);
    }
    check(coefficients.readHost(0, ordr * npts), RelationalOperator::EQUAL,
          Approx(host_knots).margin(1.0e-8), "B-spline knots of order " + std::to_string(ordr) +
          " were not computed correctly by the GPU device function.");
    
    // Compute B-spline knots and derivatives
    kCalculateBSplines<<<1, 1024>>>(dx.data(devc_layer), npts, -ordr,
                                    coefficients.data(devc_layer), derivatives.data(devc_layer));
    coefficients.download();
    derivatives.download();
    
    // Check the result against the CPU function
    std::vector<double> host_dervs(ordr * npts);
    double* hdv_ptr = host_dervs.data();
    for (int i = 0; i < npts; i++) {
      bSpline(dx.readHost(i), ordr, &hkn_ptr[i * ordr], &hdv_ptr[i * ordr]);
    }
    check(coefficients.readHost(0, ordr * npts), RelationalOperator::EQUAL,
          Approx(host_knots).margin(1.0e-8), "B-spline knots of order " + std::to_string(ordr) +
          " were not computed correctly by the GPU device function when derivatives are "
          "requested.");
    check(derivatives.readHost(0, ordr * npts), RelationalOperator::EQUAL,
          Approx(host_dervs).margin(1.0e-8), "B-spline derivatives of order " +
          std::to_string(ordr) + " were not computed correctly by the GPU device function when "
          "derivatives are requested.");
  }
}

//-------------------------------------------------------------------------------------------------
// main
//-------------------------------------------------------------------------------------------------
int main(const int argc, const char* argv[]) {

  // Some baseline initialization
  TestEnvironment oe(argc, argv);
  HpcConfig gpu_config(ExceptionResponse::WARN);
  std::vector<int> my_gpus = gpu_config.getGpuDevice(1);
  GpuDetails gpu = gpu_config.getGpuInfo(my_gpus[0]);
  
  // Section 1
  section("Vector processing capabilities");
  
  // Section 2
  section("GPU-based Xoroshiro128+ PRNG");

  // Section 3
  section("Included device functions");
  
  // Perform a summation over a double-precision real vector using the GPU
  section(1);
  const int n_tiny  = 16;
  const int n_small = 128;
  const int n_chunk = 517;
  const int n_block = 1024;
  const int n_large = 23552;
  const int n_giant = 2500000;
  Hybrid<double> tiny_set(n_tiny, "tiny_vector");
  Hybrid<double> small_set(n_small, "small_vector");
  Hybrid<double> chunk_set(n_chunk, "chunk_vector");
  Hybrid<double> block_set(n_block, "block_vector");
  Hybrid<double> large_set(n_large, "large_vector");
  Hybrid<double> giant_set(n_giant, "giant_vector");
  Hybrid<double> tb_buffer(gpu.getSMPCount(), "sum_accumulators", HybridFormat::HOST_ONLY);
  Ran2Generator prng(oe.getRandomSeed());
  loadRan2ByCPU(&tiny_set, &prng);
  loadRan2ByCPU(&small_set, &prng);
  loadRan2ByCPU(&chunk_set, &prng);
  loadRan2ByCPU(&block_set, &prng);
  tiny_set.upload();
  loadRan2ByCPU(&tiny_set, &prng);
  const double gpu_tiny_sum  = sum(tiny_set, &tb_buffer, gpu);
  const double cpu_tiny_sum  = sum(tiny_set, &tb_buffer, gpu, HybridTargetLevel::HOST);
  check(gpu_tiny_sum, RelationalOperator::EQUAL, Approx(-0.4835516929).margin(1.0e-8),
        "The tiniest vector did not sum correctly on the GPU.\n");
  check(cpu_tiny_sum, RelationalOperator::EQUAL, Approx(-1.1269689474).margin(1.0e-8),
        "The tiniest vector did not sum correctly on the CPU.\n");
  small_set.upload();
  chunk_set.upload();
  block_set.upload();
  check(sum(small_set, &tb_buffer, gpu), RelationalOperator::EQUAL, sum<double>(small_set),
        "The small vector did not sum correctly on the GPU.\n");
  check(sum(chunk_set, &tb_buffer, gpu), RelationalOperator::EQUAL, sum<double>(chunk_set),
        "The medium, odd-sized vector did not sum correctly on the GPU.\n");
  check(sum(block_set, &tb_buffer, gpu), RelationalOperator::EQUAL, sum<double>(block_set),
        "The full block-sized vector did not sum correctly on the GPU.\n");
  Xoroshiro128pGenerator fast_prng(78172);
  loadXoroshiro128pByCPU(&large_set, &fast_prng);
  loadXoroshiro128pByCPU(&giant_set, &fast_prng);
  large_set.upload();
  giant_set.upload();
  check(sum(large_set, &tb_buffer, gpu), RelationalOperator::EQUAL, sum<double>(large_set),
        "The large vector did not sum correctly on the GPU.\n");
  check(sum(giant_set, &tb_buffer, gpu), RelationalOperator::EQUAL, sum<double>(giant_set),
        "The giant vector did not sum correctly on the GPU.\n");
  
  // Test the GPU-base random number seeding and synchronized CPU/GPU generation
  const int n_cellulose_atoms = 408609;
  Hybrid<ullint2> rng128p_states(n_cellulose_atoms, "xoroshiro128p_state");
  Hybrid<ullint2> rng256pp_xy_states(n_cellulose_atoms, "xoroshiro256pp_sxy");
  Hybrid<ullint2> rng256pp_zw_states(n_cellulose_atoms, "xoroshiro256pp_szw");
  initXoroshiro128pArray(&rng128p_states, 8773925, default_xoroshiro128p_scrub, gpu);
  initXoshiro256ppArray(&rng256pp_xy_states, &rng256pp_zw_states, 4091832,
                        default_xoshiro256pp_scrub, gpu);
  std::vector<ullint2>cpu_rng128p_states(n_cellulose_atoms);
  std::vector<ullint2> cpu_rng256pp_xy_states(n_cellulose_atoms);
  std::vector<ullint2> cpu_rng256pp_zw_states(n_cellulose_atoms);
  initXoroshiro128pArray(&cpu_rng128p_states, 8773925, default_xoroshiro128p_scrub);
  initXoshiro256ppArray(&cpu_rng256pp_xy_states, &cpu_rng256pp_zw_states, 4091832,
                        default_xoshiro256pp_scrub);
  int xrs128p_deviations = 0;
  int xrs256pp_deviations = 0;
  const ullint2* rng128p_st_ptr  = rng128p_states.data();
  const ullint2* rng256pp_st_xy_ptr = rng256pp_xy_states.data();
  const ullint2* rng256pp_st_zw_ptr = rng256pp_zw_states.data();
  for (int i = 0; i < n_cellulose_atoms; i++) {
    xrs128p_deviations  += (cpu_rng128p_states[i].x != rng128p_st_ptr[i].x ||
                            cpu_rng128p_states[i].y != rng128p_st_ptr[i].y);
    xrs256pp_deviations += (cpu_rng256pp_xy_states[i].x != rng256pp_st_xy_ptr[i].x ||
                            cpu_rng256pp_xy_states[i].y != rng256pp_st_xy_ptr[i].y ||
                            cpu_rng256pp_zw_states[i].x != rng256pp_st_zw_ptr[i].x ||
                            cpu_rng256pp_zw_states[i].y != rng256pp_st_zw_ptr[i].y);
  }
  check(xrs128p_deviations, RelationalOperator::EQUAL, 0, "Deviations were found between "
        "CPU-initialized and GPU-initialized Xoroshiro128+ random number generator arrays.");
  check(xrs256pp_deviations, RelationalOperator::EQUAL, 0, "Deviations were found between "
        "CPU-initialized and GPU-initialized Xoshiro256++ random number generator arrays.");
  rng128p_states.download();
  rng256pp_xy_states.download();
  rng256pp_zw_states.download();
  Xoroshiro128pGenerator xrs128p_check(8773925);
  Xoshiro256ppGenerator xrs256pp_check(4091832);
  const int n_seeds_made = std::min(max_xo_long_jumps, n_cellulose_atoms);
  std::vector<ullint2> cpu_128p_seeds(n_seeds_made);
  std::vector<ullint2> cpu_256pp_xy_seeds(n_seeds_made);
  std::vector<ullint2> cpu_256pp_zw_seeds(n_seeds_made);
  for (int i = 0; i < n_seeds_made; i++) {
    cpu_128p_seeds[i] = xrs128p_check.revealState();
    const ullint4 cpu_found_state = xrs256pp_check.revealState();
    cpu_256pp_xy_seeds[i] = { cpu_found_state.x, cpu_found_state.y };
    cpu_256pp_zw_seeds[i] = { cpu_found_state.z, cpu_found_state.w };
    xrs128p_check.longJump();
    xrs256pp_check.longJump();
  }

  // Create a smattering of generator indices (within the bounds of the rng128p_states above) and
  // some (low) iteration counts at which to test each of them.  Predict the results on the CPU,
  // then compute them on the GPU.
  const int n_samples = 16;
  Hybrid<int2> samples(n_samples, "assorted_points");
  int2* samp_ptr = samples.data();
  for (int i = 0; i < n_samples; i++) {
    samp_ptr[i] = { 918 * i, (2 * i) + 5 };
  }
  Hybrid<double> random_output(n_samples, "random_pluckings");
  double* ro_ptr = random_output.data();
  for (int i = 0; i < n_samples; i++) {
    ro_ptr[i] = pinpointXoroshiro128p(cpu_128p_seeds, samp_ptr[i].x, samp_ptr[i].y);
  }
  samples.upload();
  const int nsmp = gpu.getSMPCount();
  const int nthr = gpu.getMaxThreadsPerBlock();
  kEvalXoroshiro128p<<<nsmp, nthr>>>(rng128p_states.data(HybridTargetLevel::DEVICE),
                                     n_cellulose_atoms, samples.data(HybridTargetLevel::DEVICE),
                                     random_output.data(HybridTargetLevel::DEVICE), n_samples);
  check(random_output.readHost(), RelationalOperator::EQUAL, random_output.readDevice(), "Random "
        "numbers from an array of Xoroshiro128+ generators computed on the CPU and GPU do not "
        "agree.");
  kEvalXoshiro256pp<<<nsmp, nthr>>>(rng256pp_xy_states.data(HybridTargetLevel::DEVICE),
                                    rng256pp_zw_states.data(HybridTargetLevel::DEVICE),
                                    n_cellulose_atoms, samples.data(HybridTargetLevel::DEVICE),
                                    random_output.data(HybridTargetLevel::DEVICE), n_samples);
  for (int i = 0; i < n_samples; i++) {
    ro_ptr[i] = pinpointXoshiro256pp(cpu_256pp_xy_seeds, cpu_256pp_zw_seeds, samp_ptr[i].x,
                                     samp_ptr[i].y);
  }
  check(random_output.readHost(), RelationalOperator::EQUAL, random_output.readDevice(), "Random "
        "numbers from an array of Xoshiro256++ generators computed on the CPU and GPU do not "
        "agree.");

  // Test other device
  section(3);
  testBSplineDeviceFuncs();
  
  // Print results
  printTestSummary(oe.getVerbosity());
  
  return 0;
}
